#include "hip/hip_runtime.h"
#include "Fix2d.h"
#include "State.h"

void __global__ compute_cu(float4 *xs, float4 *vs, float4 *fs, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        xs[idx].z = 0;
        vs[idx].z = 0;
        fs[idx].z = 0;

    }
}
//THIS NEEDS TO GO LAST


void Fix2d::compute(bool computeVirials) {
    //going to zero z in xs, vs, fs
    int nAtoms = state->atoms.size();
    GPUData &gpd = state->gpd;
    compute_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(gpd.xs.getDevData(), gpd.vs.getDevData(), gpd.fs.getDevData(), nAtoms);
    
}

void export_Fix2d() {
    boost::python::class_<Fix2d,
                          SHARED(Fix2d),
                          boost::python::bases<Fix> > (
        "Fix2d",
        boost::python::init<SHARED(State), string, int> (
            boost::python::args("state", "handle", "applyEvery"))
    )
    ;
}
