#include "hip/hip_runtime.h"
#include "Integrater.h"
#include "cuda_call.h"
// #include "globalDefs.h"

const string IntVerletType = "verlet";
const string IntRelaxType = "relax";


void Integrater::force(uint activeIdx) {
    
	int simTurn = state->turn;
	vector<Fix *> &fixes = state->fixes;
	for (Fix *f : fixes) {
		if (! (simTurn % f->applyEvery)) {
			f->compute();
		}
	}
};

void Integrater::forceSingle() {
	for (Fix *f : state->fixes) {
		if (f->forceSingle) {
			f->compute();
		}
	}
}


void Integrater::data() {
    /*
	int turn = state->turn;
	SHARED(DataManager) d = state->data;
	for (SHARED(DataSet) set : d->sets) {
		if (! ((turn - set->turnInit) % set->processEvery)) {
			set->process();
		}


	}
    */

}

void Integrater::asyncOperations() {
    int turn = state->turn;
    auto dataAndWrite = [this] (int ts) { //well, if I try to use a local state pointer, this segfaults.  Need to capture this instead.  Little confused
        for (SHARED(WriteConfig) wc : state->writeConfigs) {
            if (not ((ts - wc->turnInit) % wc->writeEvery)) {
                wc->write();
            }
        }
        for (SHARED(DataSet) ds : state->data.userSets) {
            if (not ((ts - ds->turnInit) % ds->computeEvery)) {
                ds->process(ts);
            }
        }
    };
    bool needAsync = false;
    for (SHARED(WriteConfig) wc : state->writeConfigs) {
		if (not ((turn - wc->turnInit) % wc->writeEvery)) {
            needAsync = true;
            break;
		}
    }
    if (not needAsync) {
        for (SHARED(DataSet) ds : state->data.userSets) {
            if (not ((turn - ds->turnInit) % ds->computeEvery)) {
                needAsync = true;
                break;
            }
        }
    }
    if (needAsync) {
        state->asyncHostOperation(dataAndWrite);
    }
}
/*
__global__ void printFloats(hipTextureObject_t xs, int n) {
    int idx = GETIDX();
    if (idx < n) {
        int xIdx = XIDX(idx, sizeof(float4));
        int yIdx = YIDX(idx, sizeof(float4));
        float4 x = tex2D<float4>(xs, xIdx, yIdx);
        printf("idx %d, vals %f %f %f %d\n", idx, x.x, x.y, x.z, *(int *) &x.w);

    }
}
__global__ void printFloats(float4 *xs, int n) {
    int idx = GETIDX();
    if (idx < n) {
        float4 x = xs[idx];
        printf("idx %d, vals %f %f %f %f\n", idx, x.x, x.y, x.z, x.w);

    }
}
*/


void Integrater::basicPreRunChecks() {
    if (not state->grid.isSet) {
        cout << "Atom grid is not set!" << endl;
        assert(state->grid.isSet);
    }
    if (state->rCut == RCUT_INIT) {
        cout << "rcut is not set" << endl;
        assert(state->rCut != RCUT_INIT);
    }
    if (state->is2d and state->periodic[2]) {
        cout << "2d system cannot be periodic is z dimension" << endl;
        assert(not (state->is2d and state->periodic[2]));
    }
    for (int i=0; i<3; i++) {
        if (i<2 or (i==2 and state->periodic[2])) {
            if (state->grid.ds[i] < state->rCut + state->padding) {
                cout << "Grid dimension " << i << "has discretization smaller than rCut + padding" << endl;
                assert(state->grid.ds[i] >= state->rCut + state->padding);
            }
        }
    }
    state->grid.adjustForChangedBounds();
}

void Integrater::basicPrepare(int numTurns) {
    int nAtoms = state->atoms.size();
	state->runningFor = numTurns;
    state->runInit = state->turn; 
    //Add refresh atoms!
    state->updateIdxFromIdCache(); //for updating fix atom pointers, etc
    state->prepareForRun();
    for (Fix *f : state->fixes) {
        f->updateGroupTag();
        f->prepareForRun();
    }
    for (GPUArrayBase *dat : activeData) {
        dat->dataToDevice();
    }
    state->gridGPU.periodicBoundaryConditions(state->rCut + state->padding, true);
}

void Integrater::basicFinish() {
    if (state->asyncData && state->asyncData->joinable()) {
        state->asyncData->join();
    }
    for (GPUArrayBase *dat : activeData) {
        dat->dataToHost();
    }
    hipDeviceSynchronize();
    state->downloadFromRun();

}
void Integrater::setActiveData() {
    activeData = vector<GPUArrayBase *>();
    activeData.push_back((GPUArrayBase *) &state->gpd.types);
    activeData.push_back((GPUArrayBase *) &state->gpd.xs);
    activeData.push_back((GPUArrayBase *) &state->gpd.vs);
    activeData.push_back((GPUArrayBase *) &state->gpd.fs);
    activeData.push_back((GPUArrayBase *) &state->gpd.fsLast);
    activeData.push_back((GPUArrayBase *) &state->gpd.idToIdxs);
    activeData.push_back((GPUArrayBase *) &state->gpd.qs);
}

Integrater::Integrater(State *state_, string type_) : state(state_), type(type_){
    setActiveData(); 
}

void export_Integrater() {
    class_<Integrater> ("Integrater")
        //.def("run", &Integrater::run)
        ;
}

