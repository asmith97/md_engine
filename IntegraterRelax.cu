#include "hip/hip_runtime.h"
#include "IntegraterRelax.h"
#include "cutils_func.h"
#include "State.h"


IntegraterRelax::IntegraterRelax(SHARED(State) state_) : Integrater(state_.get(), IntRelaxType) {
    //FIRE parameters
    alphaInit = 0.1;
    alphaShrink = 0.99;
    dtGrow = 1.1;
    dtShrink = 0.5;
    delay = 5;
    dtMax_mult=10;
}

//kernels for FIRE relax
//VDotF by hand
__global__ void vdotF_cu (float *dest, float4 *vs,float4 *fs, int n) {
    extern __shared__ float tmp[]; //should have length of # threads in a block (PERBLOCK)
    int potentialIdx = blockDim.x*blockIdx.x + threadIdx.x;
    if (potentialIdx < n) {
        tmp[threadIdx.x] =dot ( make_float3(vs[blockDim.x*blockIdx.x + threadIdx.x]),make_float3(fs[blockDim.x*blockIdx.x + threadIdx.x]) ) ;
    } else {
        tmp[threadIdx.x] = 0;
    }
    __syncthreads();
    int maxLookahead = log2f(blockDim.x-1);
    for (int i=0; i<=maxLookahead; i++) {
        int curLookahead = powf(2, i);
        if (! (threadIdx.x % (curLookahead*2))) {
            tmp[threadIdx.x] += tmp[threadIdx.x + curLookahead];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        atomicAdd(dest, tmp[0]);
    }
}

//update velocities
__global__ void FIRE_new_vel_cu(int nAtoms, float4 *vs, float4 *fs, float scale1, float scale2) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 vel = vs[idx];
        float4 force = fs[idx];
        float invmass = vel.w;
        float4 newVel = vel*scale1 + force*scale2;
        newVel.w = invmass;
        vs[idx] = newVel;
    }
}

//zero velocities
__global__ void zero_vel_cu(int nAtoms, float4 *vs) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 vel = vs[idx];
        vs[idx] = make_float4(0.0f,0.0f,0.0f,vel.w);
    }
}

//MD step
__global__ void FIRE_preForce_cu(int nAtoms, float4 *xs, float4 *vs, float4 *fs, float dt) {
    int idx = GETIDX();
    if (idx < nAtoms) {


        float4 vel = vs[idx];
        float4 force = fs[idx];

        float invmass = vel.w;
        float groupTag = force.w;
        xs[idx] = xs[idx] + make_float3(vel) * dt;
        float3 newVel = make_float3(force) * dt * invmass;
        vs[idx] = vel + newVel;
        fs[idx] = make_float4(0, 0, 0, groupTag);
    }
}




double IntegraterRelax::run(int numTurns, num fTol) {
    cout << "FIRE relaxation\n";
    basicPreRunChecks();  
    basicPrepare(numTurns);

    CUT_CHECK_ERROR("FIRE relaxation init failed");//Debug feature, checks error code

    //initial  values
    int lastNegative = 0;
    double dt = state->dt;
    double alpha = alphaInit;
    const double dtMax = dtMax_mult * dt;


    //assuming constant number of atoms during run
    int atomssize=state->atoms.size();
    int periodicInterval = state->periodicInterval;
    int nblock = NBLOCK(atomssize);
    int remainder = state->turn % periodicInterval;
    int turnInit = state->turn; 

    //set velocity to 0
    // 	state->gpd.vs.memsetByVal(make_float3(0.0f,0.0f,0.0f);
    zero_vel_cu <<<nblock, PERBLOCK>>>(atomssize,state->gpd.vs.getDevData());
    CUT_CHECK_ERROR("zero_vel_cu kernel execution failed");

    //vars to store kernels outputs
    GPUArray<float>VDotV(1);
    GPUArray<float>VDotF(1);
    GPUArray<float>FDotF(1);
    GPUArray<float>force(1);


    //neiblist build
    state->gridGPU.periodicBoundaryConditions(state->rCut + state->padding, true);

    for (int i=0; i<numTurns; i++) {
        //init to 0 on cpu and gpu

        VDotV.memsetByVal(0.0);
        VDotF.memsetByVal(0.0);
        FDotF.memsetByVal(0.0);
        //vdotF calc
        if (! ((remainder + i) % periodicInterval)) {
            state->gridGPU.periodicBoundaryConditions(state->rCut + state->padding, true);
        }
        asyncOperations();

        vdotF_cu <<<nblock,PERBLOCK,sizeof(float)*PERBLOCK>>>(
                    VDotF.getDevData(),
                    state->gpd.vs.getDevData(),
                    state->gpd.fs.getDevData(),
                    atomssize);
        CUT_CHECK_ERROR("vdotF_cu kernel execution failed");
        VDotF.dataToHost();

        if (VDotF.h_data[0] > 0) {

            //VdotV calc
            sumVectorSqr3D<float,float4> <<<nblock,PERBLOCK,sizeof(float)*PERBLOCK>>>(
                                            VDotV.getDevData(),
                                            state->gpd.vs.getDevData(),
                                            atomssize);
            CUT_CHECK_ERROR("vdotV_cu kernel execution failed");
            VDotV.dataToHost();

            //FdotF
            sumVectorSqr3D<float,float4> <<<nblock,PERBLOCK,sizeof(float)*PERBLOCK>>>(
                                            FDotF.getDevData(),
                                            state->gpd.fs.getDevData(),
                                            atomssize);
            CUT_CHECK_ERROR("fdotF_cu kernel execution failed");
            FDotF.dataToHost();

            float scale1 = 1 - alpha;
            float scale2 = 0;
            if (FDotF.h_data[0] != 0) {
                scale2 = alpha * sqrt(VDotV.h_data[0] / FDotF.h_data[0]);
            }
            //set velocity to
            //a.vel = a.vel * scale1 + a.force * scale2;
            FIRE_new_vel_cu <<<nblock, PERBLOCK>>>(
                                atomssize,
                                state->gpd.vs.getDevData(),
                                state->gpd.fs.getDevData(),
                                scale1,scale2);
            //check number of steps since negative 
            if (i - lastNegative > delay) {
                dt = fmin(dt*dtGrow, dtMax);
                alpha *= alphaShrink;

            }
        } else {
            lastNegative = i;
            dt *= dtShrink;
            alpha = alphaInit;
            //set velocity to 0
            //state->gpd.vs.memsetByVal(make_float3(0.0f,0.0f,0.0f);
            zero_vel_cu <<<nblock, PERBLOCK>>>(atomssize,state->gpd.vs.getDevData());
            CUT_CHECK_ERROR("zero_vel_cu kernel execution failed");

        }

        FIRE_preForce_cu <<<nblock, PERBLOCK>>>(
                            atomssize,
                            state->gpd.xs.getDevData(),
                            state->gpd.vs.getDevData(),
                            state->gpd.fs.getDevData(),
                            //state->gpd.fsLast.getDevData(),
                            dt);
        CUT_CHECK_ERROR("FIRE_preForce_cu kernel execution failed");

        int activeIdx = state->gpd.activeIdx;
        Integrater::forceSingle(activeIdx);

        if (fTol > 0 and i > delay and not (i%delay)) { //only check every so often
            //total force calc
            force.memsetByVal(0.0);

            sumVectorSqr3D<float,float4> <<<nblock,PERBLOCK,sizeof(float)*PERBLOCK>>>(
                                        force.getDevData(),
                                        state->gpd.fs.getDevData(),
                                        atomssize);
            CUT_CHECK_ERROR("kernel execution failed");//Debug feature, check error code

            force.dataToHost();
            //cout<<"Fire relax: force="<<force<<"; turns="<<i<<'\n';

            if (force.h_data[0] < fTol*fTol) {//tolerance achived, exting
                basicFinish();
                float finalForce = sqrt(force.h_data[0]);
                cout<<"FIRE relax done: force="<< finalForce <<"; turns="<<i+1<<'\n';
                return finalForce;
            }
        } 

        //shout status
        if (state->verbose and not ((state->turn - turnInit) % state->shoutEvery)) {
            cout << "Turn " << (int) state->turn << " " << (int) (100 * (state->turn - turnInit) / (num) numTurns) << " percent done" << endl;
        }
        state->turn++;

    }
    //total force calculation
    force.memsetByVal(0.0);

    sumVectorSqr3D<float,float4> <<<nblock,PERBLOCK,sizeof(float)*PERBLOCK>>>(
                                  force.getDevData(),
                                  state->gpd.fs.getDevData(),
                                  atomssize);
    CUT_CHECK_ERROR("kernel execution failed");//Debug feature, check error code

    basicFinish();

    float finalForce = sqrt(force.h_data[0]);
    cout<<"FIRE relax done: force="<< finalForce <<"; turns="<<numTurns<<'\n';
    return finalForce;
}

void export_IntegraterRelax() {
    class_<IntegraterRelax, SHARED(IntegraterRelax), bases<Integrater>, boost::noncopyable > ("IntegraterRelax", init<SHARED(State)>())
        .def("run", &IntegraterRelax::run)
        .def("set_params", &IntegraterRelax::set_params,(python::arg("alphaInit"),python::arg("alphaShrink"),python::arg("dtGrow"),python::arg("dtShrink"),python::arg("delay"),python::arg("dtMax_mult")))
        ;
}

