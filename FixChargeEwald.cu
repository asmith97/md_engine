#include "hip/hip_runtime.h"
#include "FixChargeEwald.h"
#include "cutils_func.h"
#include <hipfft/hipfft.h>
#include "cuda_call.h"
#include <fstream>

// #include <cmath>
using namespace std;

// #define THREADS_PER_BLOCK_


//different implementation for different interpolation orders
//TODO template
//order 1 nearest point
__global__ void map_charge_to_grid_order_1_cu(int nAtoms, float4 *xs,  float *qs,  BoundsGPU bounds,
                                      int3 sz,float *grid/*convert to float for cufffComplex*/) {

    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole)-bounds.lo;

        float qi = qs[idx];
        
        //find nearest grid point
        float3 h=bounds.trace()/make_float3(sz);
        int3 nearest_grid_point=make_int3(pos/h);//TODO looks unsafe. should round down
        //or
        int3 p=nearest_grid_point;
        p.x-=p.x>=sz.x? sz.x:0;
        p.y-=p.y>=sz.y? sz.y:0;
        p.z-=p.z>=sz.z? sz.z:0;
        atomicAdd(&grid[p.x*sz.y*sz.z*2+p.y*sz.z*2+p.z*2], 1.0*qi);
    }
}

inline __host__ __device__ float W_p_3(int i,float x){
    if (i==-1) return 0.125-0.5*x+0.5*x*x;
    if (i== 0) return 0.75-x*x;
    /*if (i== 1)*/ return 0.125+0.5*x+0.5*x*x;
}


__global__ void map_charge_to_grid_order_3_cu(int nAtoms, float4 *xs,  float *qs,  BoundsGPU bounds,
                                      int3 sz,float *grid/*convert to float for cufffComplex*/) {

    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole)-bounds.lo;

        float qi = qs[idx];
        
        //find nearest grid point
        float3 h=bounds.trace()/make_float3(sz);
        int3 nearest_grid_point=make_int3(pos/h);//TODO looks unsafe. should round down or handle for actual grid point where assignment happens
        
        //distance from nearest_grid_point /h
        float3 d=pos/h-make_float3(nearest_grid_point);
        
        int3 p=nearest_grid_point;
        for (int ix=-1;ix<=1;ix++){
          p.x=nearest_grid_point.x+ix;
          float charge_yz_w=qi*W_p_3(ix,d.x);
          for (int iy=-1;iy<=1;iy++){
            p.y=nearest_grid_point.y+iy;
            float charge_z_w=charge_yz_w*W_p_3(iy,d.y);
            for (int iz=-1;iz<=1;iz++){
                p.z=nearest_grid_point.z+iz;
                float charge_w=charge_z_w*W_p_3(iz,d.z);
                p.x-= p.x>=sz.x? sz.x : 0;
                p.y-= p.y>=sz.y? sz.y : 0;
                p.z-= p.z>=sz.z? sz.z : 0;
                p.x+= p.x<0    ? sz.x : 0;
                p.y+= p.y<0    ? sz.y : 0;
                p.z+= p.z<0    ? sz.z : 0;
                atomicAdd(&grid[p.x*sz.y*sz.z*2+p.y*sz.z*2+p.z*2], charge_w);
            }
          }
        }
    }
}


__global__ void map_charge_set_to_zero_cu(int3 sz,hipfftComplex *grid) {
      int3 id = make_int3( blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z*blockDim.z + threadIdx.z);

      if ((id.x<sz.x)&&(id.x<sz.y)&&(id.x<sz.z))                  
         grid[id.x*sz.y*sz.z+id.y*sz.z+id.z]=make_hipComplex (0.0f, 0.0f);    
}

__device__ float sinc(float x){
  if ((x<0.1)&&(x>-0.1)){
    float x2=x*x;
    return 1.0 - x2*0.16666666667f + x2*x2*0.008333333333333333f - x2*x2*x2*0.00019841269841269841f;    
  }
    else return sin(x)/x;
}

__global__ void Green_function_cu(BoundsGPU bounds, int3 sz,float *Green_function,float alpha,
                                  //now some parameter for Gf calc
                                  int sum_limits, int intrpl_order) {
      int3 id = make_int3( blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z*blockDim.z + threadIdx.z);

      if ((id.x<sz.x)&&(id.x<sz.y)&&(id.x<sz.z)){
          float3 h =bounds.trace()/make_float3(sz);
          
          //         2*PI
          float3 k= 6.28318530717958647693f*make_float3(id)/bounds.trace();
          if (id.x>sz.x/2) k.x= 6.28318530717958647693f*(id.x-sz.x)/bounds.trace().x;
          if (id.y>sz.y/2) k.y= 6.28318530717958647693f*(id.y-sz.y)/bounds.trace().y;
          if (id.z>sz.z/2) k.z= 6.28318530717958647693f*(id.z-sz.z)/bounds.trace().z;
          

          //OK GF(k)  = 4Pi/K^2 [SumforM(W(K+M)^2  exp(-(K+M)^2/4alpha) dot(K,K+M)/(K+M^2))] / 
          //                    [SumforM^2(W(K+M)^2)]
             
             
          float sum1=0.0f;   
          float sum2=0.0f;   
          float k2=lengthSqr(k);
          if (k2!=0.0){
              for (int ix=-sum_limits;ix<=sum_limits;ix++){//TODO different limits 
                for (int iy=-sum_limits;iy<=sum_limits;iy++){
                  for (int iz=-sum_limits;iz<=sum_limits;iz++){
                      float3 kpM=k+6.28318530717958647693f*make_float3(ix,iy,iz)/h;
//                             kpM.x+=6.28318530717958647693f/h.x*ix;//TODO rewrite
//                             kpM.y+=6.28318530717958647693f/h.y*iy;
//                             kpM.z+=6.28318530717958647693f/h.z*iz;
                            float kpMlen=lengthSqr(kpM);
                            float W=sinc(kpM.x*h.x*0.5)*sinc(kpM.y*h.y*0.5)*sinc(kpM.z*h.z*0.5);
                            for(int p=1;p<intrpl_order;p++)
                                  W*=W;
    //                          W*=h;//not need- cancels out
                            float W2=W*W;
                            
                            //4*PI
                            sum1+=12.56637061435917295385*exp(-kpMlen*0.25/alpha/alpha)*dot(k,kpM)/kpMlen*W2;
                            sum2+=W2;
                  }
                }
              }
              Green_function[id.x*sz.y*sz.z+id.y*sz.z+id.z]=sum1/(sum2*sum2)/k2;
          }else{
              Green_function[id.x*sz.y*sz.z+id.y*sz.z+id.z]=0.0f;
          }
      }
             
}

__global__ void potential_cu(int3 sz,float *Green_function,
                                    hipfftComplex *FFT_qs, hipfftComplex *FFT_phi){
      int3 id = make_int3( blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z*blockDim.z + threadIdx.z);

      if ((id.x<sz.x)&&(id.x<sz.y)&&(id.x<sz.z)){
        FFT_phi[id.x*sz.y*sz.z+id.y*sz.z+id.z]=FFT_qs[id.x*sz.y*sz.z+id.y*sz.z+id.z]*Green_function[id.x*sz.y*sz.z+id.y*sz.z+id.z];
//TODO after Inverse FFT divide by volume
      }
}

__global__ void E_field_cu(BoundsGPU bounds, int3 sz,float *Green_function, hipfftComplex *FFT_qs,
                           hipfftComplex *FFT_Ex,hipfftComplex *FFT_Ey,hipfftComplex *FFT_Ez){
      int3 id = make_int3( blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z*blockDim.z + threadIdx.z);

      if ((id.x<sz.x)&&(id.x<sz.y)&&(id.x<sz.z)){
          //K vector
          float3 k= 6.28318530717958647693f*make_float3(id)/bounds.trace();
          if (id.x>sz.x/2) k.x= 6.28318530717958647693f*(id.x-sz.x)/bounds.trace().x;
          if (id.y>sz.y/2) k.y= 6.28318530717958647693f*(id.y-sz.y)/bounds.trace().y;
          if (id.z>sz.z/2) k.z= 6.28318530717958647693f*(id.z-sz.z)/bounds.trace().z;        
          
          //ik*q(k)*Gf(k)
          hipfftComplex Ex,Ey,Ez;
          float GF=Green_function[id.x*sz.y*sz.z+id.y*sz.z+id.z];
          hipfftComplex q=FFT_qs[id.x*sz.y*sz.z+id.y*sz.z+id.z];

          Ex.y= k.x*q.x*GF;
          Ex.x=-k.x*q.y*GF;
          Ey.y= k.y*q.x*GF;
          Ey.x=-k.y*q.y*GF;
          Ez.y= k.z*q.x*GF;
          Ez.x=-k.z*q.y*GF;
          
          FFT_Ex[id.x*sz.y*sz.z+id.y*sz.z+id.z]=Ex;
          FFT_Ey[id.x*sz.y*sz.z+id.y*sz.z+id.z]=Ey;
          FFT_Ez[id.x*sz.y*sz.z+id.y*sz.z+id.z]=Ez;
          //TODO after Inverse FFT divide by -volume
      }
}


__global__ void Ewald_long_range_forces_order_1_cu(int nAtoms, float4 *xs, float4 *fs, 
                                                   float *qs, BoundsGPU bounds,
                                                   int3 sz, hipfftComplex *FFT_Ex,
                                                    hipfftComplex *FFT_Ey,hipfftComplex *FFT_Ez){
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole)-bounds.lo;

        float qi = qs[idx];
        
        //find nearest grid point
        float3 h=bounds.trace()/make_float3(sz);
        int3 p=make_int3(pos/h);//TODO looks unsafe. should round down
        p.x-=p.x>=sz.x? sz.x:0;
        p.y-=p.y>=sz.y? sz.y:0;
        p.z-=p.z>=sz.z? sz.z:0;
        
        //get E field
        float3 E;
        float volume=bounds.trace().x*bounds.trace().y*bounds.trace().z;
        E.x= -FFT_Ex[p.x*sz.y*sz.z+p.y*sz.z+p.z].x/volume;
        E.y= -FFT_Ey[p.x*sz.y*sz.z+p.y*sz.z+p.z].x/volume;
        E.z= -FFT_Ez[p.x*sz.y*sz.z+p.y*sz.z+p.z].x/volume;
        
        float3 force=qi*E;
        fs[idx] += force;
    }
}


__global__ void Ewald_long_range_forces_order_3_cu(int nAtoms, float4 *xs, float4 *fs, 
                                                   float *qs, BoundsGPU bounds,
                                                   int3 sz, hipfftComplex *FFT_Ex,
                                                    hipfftComplex *FFT_Ey,hipfftComplex *FFT_Ez){
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole)-bounds.lo;

        float qi = qs[idx];
        

        //find nearest grid point
        float3 h=bounds.trace()/make_float3(sz);
        int3 nearest_grid_point=make_int3(pos/h);//TODO looks unsafe. should round down or handle for actual grid point where assignment happens
        
        //distance from nearest_grid_point /h
        float3 d=pos/h-make_float3(nearest_grid_point);

        float3 E=make_float3(0,0,0);
        float volume=bounds.trace().x*bounds.trace().y*bounds.trace().z;

        int3 p=nearest_grid_point;
        for (int ix=-1;ix<=1;ix++){
          p.x=nearest_grid_point.x+ix;
          for (int iy=-1;iy<=1;iy++){
            p.y=nearest_grid_point.y+iy;
            for (int iz=-1;iz<=1;iz++){
                p.z=nearest_grid_point.z+iz;
                p.x-= p.x>=sz.x? sz.x : 0;
                p.y-= p.y>=sz.y? sz.y : 0;
                p.z-= p.z>=sz.z? sz.z : 0;
                p.x+= p.x<0    ? sz.x : 0;
                p.y+= p.y<0    ? sz.y : 0;
                p.z+= p.z<0    ? sz.z : 0;
                float3 Ep;
                float W_xyz=W_p_3(ix,d.x)*W_p_3(iy,d.y)*W_p_3(iz,d.z);
                
                Ep.x= -FFT_Ex[p.x*sz.y*sz.z+p.y*sz.z+p.z].x/volume;
                Ep.y= -FFT_Ey[p.x*sz.y*sz.z+p.y*sz.z+p.z].x/volume;
                Ep.z= -FFT_Ez[p.x*sz.y*sz.z+p.y*sz.z+p.z].x/volume;
                E+=W_xyz*Ep;
            }
          }
        }
               
        float3 force=qi*E;
        fs[idx] += force;
    }
}


__global__ void compute_short_range_forces_cu(int nAtoms, float4 *xs, float4 *fs, int *neighborCounts, uint *neighborlist, int *cumulSumMaxPerBlock, float *qs, float alpha, float rCut, BoundsGPU bounds, int warpSize, float oneFourStrength) {

    float multipliers[4] = {1, 0, 0, oneFourStrength};
    int idx = GETIDX();
    if (idx < nAtoms) {
        float4 posWhole = xs[idx];
        float3 pos = make_float3(posWhole);

        float3 forceSum = make_float3(0, 0, 0);
        float qi = qs[idx];//tex2D<float>(qs, XIDX(idx, sizeof(float)), YIDX(idx, sizeof(float)));

        //printf("start, end %d %d\n", start, end);
        int baseIdx = baseNeighlistIdx<void>(cumulSumMaxPerBlock, warpSize);
        int numNeigh = neighborCounts[idx];
        for (int i=0; i<numNeigh; i++) {
            int nlistIdx = baseIdx + warpSize * i;
            uint otherIdxRaw = neighborlist[nlistIdx];
            uint neighDist = otherIdxRaw >> 30;
            uint otherIdx = otherIdxRaw & EXCL_MASK;
            float3 otherPos = make_float3(xs[otherIdx]);
            //then wrap and compute forces!
            float3 dr = bounds.minImage(pos - otherPos);
            float lenSqr = lengthSqr(dr);
            //   printf("dist is %f %f %f\n", dr.x, dr.y, dr.z);
            if (lenSqr < rCut*rCut) {
                float multiplier = multipliers[neighDist];
                float len=sqrtf(lenSqr);
                float qj = qs[otherIdx];

                float r2inv = 1.0f/lenSqr;
                float rinv = 1.0f/len;                                   //1/Sqrt(Pi)
                float forceScalar = qi*qj*(erfcf((alpha*len))*rinv+(2.0*0.5641895835477563*alpha)*exp(-alpha*alpha*lenSqr))*r2inv* multiplier;

                
                float3 forceVec = dr * forceScalar;
                forceSum += forceVec;
            }

        }   
        fs[idx] += forceSum; //operator for float4 + float3

    }

}


FixChargeEwald::FixChargeEwald(SHARED(State) state_, string handle_, string groupHandle_): FixCharge(state_, handle_, groupHandle_, chargePairDSF),first_run(true){
//   setParameters(128,3.0);
  hipfftCreate(&plan);
}


FixChargeEwald::~FixChargeEwald(){
  hipfftDestroy(plan);
  hipFree(FFT_Qs);
  hipFree(FFT_Ex);
  hipFree(FFT_Ey);
  hipFree(FFT_Ez);
}


void FixChargeEwald::setParameters(int szx_,int szy_,int szz_,float rcut_,int interpolation_order_)
{
    //for now support for only 2^N sizes
    //TODO generalize for non cubic boxes
    if ((szx_!=32)||(szx_!=64)||(szx_!=128)||(szx_!=256)||(szx_!=512)||(szx_!=1024)){
        cout << szx_ << " is not supported, sorry. Only 2^N grid size works for charge Ewald\n";
    }
    if ((szy_!=32)||(szy_!=64)||(szy_!=128)||(szy_!=256)||(szy_!=512)||(szy_!=1024)){
        cout << szy_ << " is not supported, sorry. Only 2^N grid size works for charge Ewald\n";
    }
    if ((szz_!=32)||(szz_!=64)||(szz_!=128)||(szz_!=256)||(szz_!=512)||(szz_!=1024)){
        cout << szz_ << " is not supported, sorry. Only 2^N grid size works for charge Ewald\n";
    }
    sz=make_int3(szx_,szy_,szz_);
    r_cut=rcut_;
    hipMalloc((void**)&FFT_Qs, sizeof(hipfftComplex)*sz.x*sz.y*sz.z);

    hipfftPlan3d(&plan, sz.x,sz.y, sz.z, HIPFFT_C2C);

    
    hipMalloc((void**)&FFT_Ex, sizeof(hipfftComplex)*sz.x*sz.y*sz.z);
    hipMalloc((void**)&FFT_Ey, sizeof(hipfftComplex)*sz.x*sz.y*sz.z);
    hipMalloc((void**)&FFT_Ez, sizeof(hipfftComplex)*sz.x*sz.y*sz.z);
    
    Green_function=GPUArray<float>(sz.x*sz.y*sz.z);
    CUT_CHECK_ERROR("setParameters execution failed");
    

    interpolation_order=interpolation_order_;
    //in order to find alpha we have to solve
    //Fshort(r_cut,alpha)==10^-10
    //where Fshort(r,alpha)= erfc(alpha*r)/r^2+2alpha/sqrt(pi)*exp(-alpha^2*r^2)/r
    
    //first we solve with only the  leading term exp(-alpha^2*r_cut^2)====10^-10
    //which gives us  alpha=4.79853/r_cut
    alpha=4.79853/r_cut;
    //second TODO couple of iterations of Newton root finder
    cout<<"Ewald alpha="<<alpha<<'\n';
}


void FixChargeEwald::calc_Green_function(){

    
    dim3 dimBlock(8,8,8);
    dim3 dimGrid((sz.x + dimBlock.x - 1) / dimBlock.x,(sz.y + dimBlock.y - 1) / dimBlock.y,(sz.z + dimBlock.z - 1) / dimBlock.z);    
    Green_function_cu<<<dimGrid, dimBlock>>>(state->boundsGPU, sz,Green_function.getDevData(),alpha,
                                             10,interpolation_order);//TODO parameters unknown
    CUT_CHECK_ERROR("Green_function_cu kernel execution failed");
    
        //test area
//     Green_function.dataToHost();
//     ofstream ofs;
//     ofs.open("test_Green_function.dat",ios::out );
//     for(int i=0;i<sz.x;i++)
//             for(int j=0;j<sz.y;j++){
//                 for(int k=0;k<sz.z;k++){
//                     cout<<Green_function.h_data[i*sz.y*sz.z+j*sz.z+k]<<'\t';
//                     ofs<<Green_function.h_data[i*sz.y*sz.z+j*sz.z+k]<<'\t';
//                 }
//                 ofs<<'\n';
//                 cout<<'\n';
//             }
//     ofs.close();

}


void FixChargeEwald::calc_potential(hipfftComplex *phi_buf){
     Bounds b=state->bounds;
    float volume=b.trace[0]*b.trace[1]*b.trace[2];
    
    dim3 dimBlock(8,8,8);
    dim3 dimGrid((sz.x + dimBlock.x - 1) / dimBlock.x,(sz.y + dimBlock.y - 1) / dimBlock.y,(sz.z + dimBlock.z - 1) / dimBlock.z);    
    potential_cu<<<dimGrid, dimBlock>>>(sz,Green_function.getDevData(), FFT_Qs,phi_buf);
    CUT_CHECK_ERROR("potential_cu kernel execution failed");    


    hipfftExecC2C(plan, phi_buf, phi_buf,  HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("hipfftExecC2C execution failed");

    //test area
    float *buf=new float[sz.x*sz.y*sz.z*2];
    hipMemcpy((void *)buf,phi_buf,sizeof(hipfftComplex)*sz.x*sz.y*sz.z,hipMemcpyDeviceToHost );
    ofstream ofs;
    ofs.open("test_phi.dat",ios::out );
    for(int i=0;i<sz.x;i++)
            for(int j=0;j<sz.y;j++){
                for(int k=0;k<sz.z;k++){
                    cout<<buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                    ofs<<buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                }
                ofs<<'\n';
                cout<<'\n';
            }
    ofs.close();
    delete []buf;
}



void FixChargeEwald::compute() {
    CUT_CHECK_ERROR("before FixChargeEwald kernel execution failed");

    cout<<"FixChargeEwald::compute..\n";
    int nAtoms = state->atoms.size();
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx;
    int *neighborCounts = grid.perAtomArray.d_data.ptr;
    
    if (first_run){
        first_run=false;
        calc_Green_function();
    }
    
 

    //first update grid from atoms positions
    //set qs to 0
    dim3 dimBlock(8,8,8);
    dim3 dimGrid((sz.x + dimBlock.x - 1) / dimBlock.x,(sz.y + dimBlock.y - 1) / dimBlock.y,(sz.z + dimBlock.z - 1) / dimBlock.z);    
    map_charge_set_to_zero_cu<<<dimGrid, dimBlock>>>(sz,FFT_Qs);
    
      switch (interpolation_order){
      case 1:{map_charge_to_grid_order_1_cu
              <<<NBLOCK(nAtoms), PERBLOCK>>>( nAtoms,
                                              gpd.xs(activeIdx),                                                      
                                              gpd.qs(activeIdx),
                                              state->boundsGPU,
                                              sz,
                                              (float *)FFT_Qs);
              break;}
      case 3:{map_charge_to_grid_order_3_cu
              <<<NBLOCK(nAtoms), PERBLOCK>>>( nAtoms,
                                              gpd.xs(activeIdx),                                                      
                                              gpd.qs(activeIdx),
                                              state->boundsGPU,
                                              sz,
                                              (float *)FFT_Qs);
              break;}
    }    
    CUT_CHECK_ERROR("map_charge_to_grid_cu kernel execution failed");

    hipfftExecC2C(plan, FFT_Qs, FFT_Qs, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("hipfftExecC2C Qs execution failed");

    
//     //test area
//     float buf[sz.x*sz.y*sz.z*2];
//     hipMemcpy(buf,FFT_Qs,sizeof(hipfftComplex)*sz.x*sz.y*sz.z,hipMemcpyDeviceToHost );
//     ofstream ofs;
//     ofs.open("test_FFT.dat",ios::out );
//     for(int i=0;i<sz.x;i++)
//             for(int j=0;j<sz.y;j++){
//                 for(int k=0;k<sz.z;k++){
//                     cout<<buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]<<'\t';
//                     ofs <<buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]<<'\t';
//                 }
//                 ofs<<'\n';
//                 cout<<'\n';
//             }
//     ofs.close();

    
    //next potential calculation: just going to use Ex to store it for now
//       calc_potential(FFT_Ex);

    //calc E field
    E_field_cu<<<dimGrid, dimBlock>>>(state->boundsGPU,sz,Green_function.getDevData(), FFT_Qs,FFT_Ex,FFT_Ey,FFT_Ez);
    CUT_CHECK_ERROR("E_field_cu kernel execution failed");    


    hipfftExecC2C(plan, FFT_Ex, FFT_Ex,  HIPFFT_BACKWARD);
    hipfftExecC2C(plan, FFT_Ey, FFT_Ey,  HIPFFT_BACKWARD);
    hipfftExecC2C(plan, FFT_Ez, FFT_Ez,  HIPFFT_BACKWARD);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("hipfftExecC2C  E_field execution failed");
    
    
    /*//test area
     Bounds b=state->bounds;
    float volume=b.trace[0]*b.trace[1]*b.trace[2];    
    float *buf=new float[sz.x*sz.y*sz.z*2];
    hipMemcpy((void *)buf,FFT_Ex,sizeof(hipfftComplex)*sz.x*sz.y*sz.z,hipMemcpyDeviceToHost );
    ofstream ofs;
    ofs.open("test_Ex.dat",ios::out );
    for(int i=0;i<sz.x;i++)
            for(int j=0;j<sz.y;j++){
                for(int k=0;k<sz.z;k++){
                    cout<<-buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                    ofs<<-buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                }
                ofs<<'\n';
                cout<<'\n';
            }
    ofs.close();
    hipMemcpy((void *)buf,FFT_Ey,sizeof(hipfftComplex)*sz.x*sz.y*sz.z,hipMemcpyDeviceToHost );
    ofs.open("test_Ey.dat",ios::out );
    for(int i=0;i<sz.x;i++)
            for(int j=0;j<sz.y;j++){
                for(int k=0;k<sz.z;k++){
                    cout<<-buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                    ofs<<-buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                }
                ofs<<'\n';
                cout<<'\n';
            }
    ofs.close();    
    hipMemcpy((void *)buf,FFT_Ez,sizeof(hipfftComplex)*sz.x*sz.y*sz.z,hipMemcpyDeviceToHost );
    ofs.open("test_Ez.dat",ios::out );
    for(int i=0;i<sz.x;i++)
            for(int j=0;j<sz.y;j++){
                for(int k=0;k<sz.z;k++){
                    cout<<-buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                    ofs<<-buf[i*sz.y*sz.z*2+j*sz.z*2+k*2]/volume<<'\t';
                }
                ofs<<'\n';
                cout<<'\n';
            }
    ofs.close();    
    delete []buf;   */ 
    
    
    //calc forces
    switch (interpolation_order){
      case 1:{Ewald_long_range_forces_order_1_cu<<<NBLOCK(nAtoms), PERBLOCK>>>( nAtoms,
                                              gpd.xs(activeIdx),                                                      
                                              gpd.fs(activeIdx),
                                              gpd.qs(activeIdx),
                                              state->boundsGPU,
                                              sz,
                                              FFT_Ex,FFT_Ey,FFT_Ez);
              break;}
      case 3:{Ewald_long_range_forces_order_3_cu<<<NBLOCK(nAtoms), PERBLOCK>>>( nAtoms,
                                              gpd.xs(activeIdx),                                                      
                                              gpd.fs(activeIdx),
                                              gpd.qs(activeIdx),
                                              state->boundsGPU,
                                              sz,
                                              FFT_Ex,FFT_Ey,FFT_Ez);
               break;}
    }
    CUT_CHECK_ERROR("Ewald_long_range_forces_cu  execution failed");
    
    
    compute_short_range_forces_cu<<<NBLOCK(nAtoms), PERBLOCK>>>( nAtoms,
                                              gpd.xs(activeIdx),                                                      
                                              gpd.fs(activeIdx),
                                              neighborCounts,
                                              grid.neighborlist.ptr,
                                              grid.perBlockArray.d_data.ptr,
                                              gpd.qs(activeIdx),
                                              alpha,
                                              r_cut,
                                              state->boundsGPU,
                                              state->devManager.prop.warpSize, 0.5);
    CUT_CHECK_ERROR("Ewald_short_range_forces_cu  execution failed");
    
}


void export_FixChargeEwald() {
//     class_<FixChargeEwald, SHARED(FixChargeEwald), bases<FixCharge> > ("FixChargeEwald", init<SHARED(State), string, string> (args("state", "handle", "groupHandle")))
//         .def("setParameters", &FixChargeEwald::setParameters, (python::arg("alpha"), python::arg("r_cut")))
//         ;
}
