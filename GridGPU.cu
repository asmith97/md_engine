#include "hip/hip_runtime.h"
#include "GridGPU.h"
#include <set>
#include "State.h"
#include "helpers.h"
#include "Bond.h"
#include "BoundsGPU.h"
#include "list_macro.h"
#include "Mod.h"
#include "Fix.h"
#include "cutils_func.h"
//for debugging
__global__ void countNumInGridCells(hipTextureObject_t xs, int nAtoms, int *counts, int *atomIdxs, float3 os, float3 ds, int3 ns) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        //printf("idx %d\n", idx);
        int xIdx = XIDX(idx, sizeof(float4));
        int yIdx = YIDX(idx, sizeof(float4));
        int3 sqrIdx = make_int3((make_float3(tex2D<float4>(xs, xIdx, yIdx)) - os) / ds);
        int sqrLinIdx = LINEARIDX(sqrIdx, ns);
        //printf("lin is %d\n", sqrLinIdx);
        int myPlaceInGrid = atomicAdd(counts + sqrLinIdx, 1); //atomicAdd returns old value
        //printf("grid is %d\n", myPlaceInGrid);
        //printf("myPlaceInGrid %d\n", myPlaceInGrid);
        atomIdxs[idx] = myPlaceInGrid;
        //okay - atoms seem to be getting assigned the right idx in grid 
    }
}


__global__ void periodicWrap(hipSurfaceObject_t xs, int nAtoms, BoundsGPU bounds) {
    int idx = GETIDX();
    if (idx < nAtoms) {

        int xIdx = XIDX(idx, sizeof(float4));
        int yIdx = YIDX(idx, sizeof(float4));
        int xAddr = xIdx * sizeof(float4);
        float4 pos = surf2Dread<float4>(xs, xAddr, yIdx);
        float4 orig = pos;
        float id = pos.w;
        float3 trace = bounds.trace();
        float3 diffFromLo = make_float3(pos) - bounds.lo;
        float3 imgs = floorf(diffFromLo / trace); //are unskewed at this point
        float3 pos_orig = make_float3(pos);
        pos -= make_float4(trace * imgs * bounds.periodic);
        pos.w = id;
        if (not(pos.x==orig.x and pos.y==orig.y and pos.z==orig.z)) { //sigh
            surf2Dwrite(pos, xs, xAddr, yIdx);
        }

    }

}
#define TESTIDX 1000
/*
__global__ void printFloats(hipTextureObject_t xs, int n) {
    int idx = GETIDX();
    if (idx < n) {
        int xIdx = XIDX(idx);
        int yIdx = YIDX(idx);
        float4 x = tex2D<float4>(xs, xIdx, yIdx);
        printf("idx %d, vals %f %f %f\n", idx, x.x, x.y, x.z);

    }
}

__global__ void printInts(hipTextureObject_t xs, int n) {
    int idx = GETIDX();
    if (idx < n) {
        int xIdx = XIDX(idx);
        int yIdx = YIDX(idx);
        int x = tex2D<int>(xs, xIdx, yIdx);
        printf("idx %d, val %d\n", idx, x);

    }
}


__global__ void printIntsArray(int *xs, int n) {
    int idx = GETIDX();
    if (idx < n) {
        int x = xs[idx];
        printf("idx %d, val %d\n", idx, x);
    }
}


__global__ void printNeighbors(int *neighborlistBounds, hipTextureObject_t neighbors, int nAtoms) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        int begin = neighborlistBounds[idx];
        int end = neighborlistBounds[idx+1];
        for (int i=begin; i<end; i++) {
            int xIdx = XIDX(i);
            int yIdx = YIDX(i);
            int x = tex2D<int>(neighbors, xIdx, yIdx);
            printf("idx %d has neighbor of idx %d\n", idx, x);
        }
    }
}
*/
template <typename T>
__device__ void copyToOtherSurf(hipSurfaceObject_t from, hipSurfaceObject_t to, int idx_init, int idx_final) {
    int xIdx, yIdx, xAddr;
    xIdx = XIDX(idx_init, sizeof(T));
    yIdx = YIDX(idx_init, sizeof(T));
    xAddr = xIdx * sizeof(T);
    T val = surf2Dread<T>(from, xAddr, yIdx);
    xIdx = XIDX(idx_final, sizeof(T));
    yIdx = YIDX(idx_final, sizeof(T));
    xAddr = xIdx * sizeof(T);
    surf2Dwrite(val, to, xAddr, yIdx);
}

template <typename T>
__device__ void copyToOtherList(T *from, T *to, int idx_init, int idx_final) {
    to[idx_final] = from[idx_init];
}

__global__ void sortPerAtomArrays(
        hipSurfaceObject_t xsFrom, hipSurfaceObject_t xsTo, 
        float4  *vsFrom,        float4  *vsTo,
        float4  *fsFrom,        float4  *fsTo,
        float4  *fsLastFrom,    float4  *fsLastTo,
        hipSurfaceObject_t typesFrom, hipSurfaceObject_t typesTo,
        hipSurfaceObject_t qsFrom, hipSurfaceObject_t qsTo,

        hipSurfaceObject_t idToIdx,
        int *gridCellArrayIdxs, int *idxInGridCell, int nAtoms, float3 os, float3 ds, int3 ns) {

    int idx = GETIDX();
    if (idx < nAtoms) {
        int xIdx = XIDX(idx, sizeof(float4));
        int yIdx = YIDX(idx, sizeof(float4));
        int xAddr = xIdx * sizeof(float4);
        float4 posWhole = surf2Dread<float4>(xsFrom, xAddr, yIdx);
        int id = * (int *) &posWhole.w;
        float3 pos = make_float3(posWhole);
        int3 sqrIdx = make_int3((pos - os) / ds);
        int sqrLinIdx = LINEARIDX(sqrIdx, ns);
        int sortedIdx = gridCellArrayIdxs[sqrLinIdx] + idxInGridCell[idx];

        //okay, now have all data needed to do copies
        copyToOtherSurf<float4>(xsFrom, xsTo, idx, sortedIdx);
        copyToOtherSurf<short>(typesFrom, typesTo, idx, sortedIdx);
        copyToOtherList<float4>(vsFrom, vsTo, idx, sortedIdx);
        copyToOtherList<float4>(fsFrom, fsTo, idx, sortedIdx);
        copyToOtherList<float4>(fsLastFrom, fsLastTo, idx, sortedIdx);
        copyToOtherSurf<float>(qsFrom, qsTo, idx, sortedIdx);

        int xAddrId = XIDX(id, sizeof(int)) * sizeof(int);
        int yIdxId = YIDX(id, sizeof(int));

        surf2Dwrite(sortedIdx, idToIdx, xAddrId, yIdxId);

    //annnnd copied!


        




    }
}


//        gridNonSort<<<NBLOCK(nAtoms), PERBLOCK>>>(state->gpd.xs.tex[activeIdx], state->gpd.xs.surf[!activeIdx], state->gpd.ids(activeIdx), state->gpd.ids(!activeIdx), nAtoms, perCellArray.ptr, perAtomArray.ptr, os, ds, ns);
__global__ void gridNonSort(hipTextureObject_t xs, hipSurfaceObject_t xsGrid, int nAtoms, int *gridCellArrayIdxs, int *idxInGridCell, float3 os, float3 ds, int3 ns) {
    int idx = GETIDX();
    if (idx < nAtoms) {
        int xIdx = XIDX(idx, sizeof(float4));
        int yIdx = YIDX(idx, sizeof(float4));
        float4 posWhole = tex2D<float4>(xs, xIdx, yIdx);
        float3 pos = make_float3(posWhole);
        int3 sqrIdx = make_int3((pos - os) / ds);
        int sqrLinIdx = LINEARIDX(sqrIdx, ns); //only uses xyz
        int sortedIdx = gridCellArrayIdxs[sqrLinIdx] + idxInGridCell[idx];

        xIdx = XIDX(sortedIdx, sizeof(float4));
        yIdx = YIDX(sortedIdx, sizeof(float4));
        int xAddr = xIdx * sizeof(float4);
        surf2Dwrite(posWhole, xsGrid, xAddr, yIdx); //id is carried along with this

    }
}
/*
void gridCPU(GPUArrayTexPair<float4> &xs, int activeIdx, int nAtoms, GPUArrayDevice<int> &perCellArray, GPUArrayDevice<int> &perAtomArray, float3 os, float3 ds, int3 ns) {

    set<int> sortedAtoms;
    xs.dataToHost();
    int *gridCellArrayIdxs = perCellArray.get((int *) NULL);
    int *idxInGridCell = perAtomArray.get((int *) NULL);
    hipDeviceSynchronize();
    for (int i=0; i<nAtoms; i++) {
        float4 posWhole = xs.h_data[i];
        float3 pos = make_float3(posWhole);
        if (pos.x < os.x or pos.y < os.y or pos.z < os.z or pos.x>=os.x+(ds.x*ns.x) or pos.y>=os.y+(ds.y*ns.y) or pos.z>=os.z+(ds.z*ns.z)) {
            if (fabs(pos.x) != 0 and fabs(pos.y) != 0) {
                cout << fabs
                cout << "Bad position " << Vector(pos) << endl;
            }
        }
        int3 sqrIdx = make_int3((pos-os)/ds);
        int sqrLinIdx = LINEARIDX(sqrIdx, ns);
        if (sqrLinIdx < 0 or sqrLinIdx >= perCellArray.n) {
            cout << "bad cell array index " << sqrLinIdx << endl;
            cout << "my pos is " << Vector(pos) << endl;
        }
        int sortedIdx = gridCellArrayIdxs[sqrLinIdx] + idxInGridCell[i];
        if (sortedIdx < 0 or sortedIdx >= nAtoms) {
            cout << "sorted index out of bounds!" << endl;
        }
        auto inserted = sortedAtoms.insert(sortedIdx);
        if (!inserted.second) {
            cout << "duplicate index!" << endl;
        }

    }
    free(gridCellArrayIdxs);
    free(idxInGridCell);

}
*/

__device__ bool isExcluded(const int id, int *exclusions, const int numExclusions, const int maxExclusions) { //exclusions should be shared memory or this will be just silly-slow
    for (int i=0; i<numExclusions; i++) {
        if (id == exclusions[maxExclusions * threadIdx.x + i]) {
            return true;
        }
    }
    return false;
}

__device__ void checkCell(float3 pos, int idx, int myId, int myIdx, hipTextureObject_t xs, int *neighborCounts, int *gridCellArrayIdxs, hipTextureObject_t idToIdxs, int squareIdx, float3 offset, float3 trace, float neighCutSqr) {//, int *exclusions, int numExclusions, int maxExclusions) {
    int idxMin = gridCellArrayIdxs[squareIdx];
    int idxMax = gridCellArrayIdxs[squareIdx+1];
    float3 loop = offset * trace;
    for (int i=idxMin; i<idxMax; i++) {
        int xIdx = XIDX(i, sizeof(float4));
        int yIdx = YIDX(i, sizeof(float4));
        float4 otherPosWhole = tex2D<float4>(xs, xIdx, yIdx); 
        float3 otherPos = make_float3(otherPosWhole);
        float3 distVec = otherPos + loop - pos;
        int otherId = *(int *) &otherPosWhole.w;
        if (otherId != myId && dot(distVec, distVec) < neighCutSqr /*&& !(isExcluded(otherId, exclusions, numExclusions, maxExclusions))*/) {
            neighborCounts[myIdx] ++;

        }

    }
}
__global__ void countNumNeighbors(hipTextureObject_t xs, int nAtoms, hipTextureObject_t idToIdxs, int *neighborCounts, int *gridCellArrayIdxs, float3 os, float3 ds, int3 ns, float3 periodic, float3 trace, float neighCutSqr, bool justSorted/*, hipTextureObject_t exclusionIdxs, hipTextureObject_t exclusions, int maxExclusions*/) {

   // extern __shared__ int exclusions_shr[]; 
    int idx = GETIDX();
    if (idx < nAtoms) {
        int xIdxAtom = XIDX(idx, sizeof(float4));
        int yIdxAtom = YIDX(idx, sizeof(float4));
        float4 posWhole = tex2D<float4>(xs, xIdxAtom, yIdxAtom);
        int myId = *(int *)&posWhole.w;


        /*int exclIdxLo = tex2D<int>(exclusionIdxs, XIDX(myId, sizeof(int)), YIDX(myId, sizeof(int)));
        int exclIdxHi = tex2D<int>(exclusionIdxs, XIDX(myId+1, sizeof(int)), YIDX(myId+1, sizeof(int)));
        int numExclusions = exclIdxHi - exclIdxLo;
        for (int i=0; i<numExclusions; i++) {
            exclusions_shr[threadIdx.x*maxExclusions + i] = tex2D<int>(exclusions, XIDX(exclIdxLo + i, sizeof(int)), YIDX(exclIdxLo + i, sizeof(int)));
        }
*/
        float3 pos = make_float3(posWhole);
        int3 sqrIdx = make_int3((pos - os) / ds);
        int xIdx, yIdx, zIdx;
        int xIdxLoop, yIdxLoop, zIdxLoop;
        float3 offset = make_float3(0, 0, 0);
        int myIdx;
        if (justSorted) {
            myIdx = idx;
        } else {
            int xIdxID = XIDX(myId, sizeof(int));
            int yIdxID = YIDX(myId, sizeof(int));
            myIdx = tex2D<int>(idToIdxs, xIdxID, yIdxID);
        }


        for (xIdx=sqrIdx.x-1; xIdx<=sqrIdx.x+1; xIdx++) {
            offset.x = -floorf((float) xIdx / ns.x);
            xIdxLoop = xIdx + ns.x * offset.x;
       
            if (periodic.x || (!periodic.x && xIdxLoop == xIdx)) {

                for (yIdx=sqrIdx.y-1; yIdx<=sqrIdx.y+1; yIdx++) {
                    offset.y = -floorf((float) yIdx / ns.y);
                    yIdxLoop = yIdx + ns.y * offset.y;
                    if (periodic.y || (!periodic.y && yIdxLoop == yIdx)) {

                        for (zIdx=sqrIdx.z-1; zIdx<=sqrIdx.z+1; zIdx++) {
                            offset.z = -floorf((float) zIdx / ns.z);
                            zIdxLoop = zIdx + ns.z * offset.z;
                            if (periodic.z || (!periodic.z && zIdxLoop == zIdx)) {
                                int3 sqrIdxOther = make_int3(xIdxLoop, yIdxLoop, zIdxLoop);
                                int sqrIdxOtherLin = LINEARIDX(sqrIdxOther, ns);
                                checkCell(pos, idx, myId, myIdx, xs, neighborCounts, gridCellArrayIdxs, idToIdxs, sqrIdxOtherLin, -offset, trace, neighCutSqr);//, exclusions_shr, numExclusions, maxExclusions);
                                //note sign switch on offset!

                            }
                        }
                    }
                }


            }
        }
    }
}


__device__ uint addExclusion(uint otherId, uint *exclusionIds_shr, int idxLo, int idxHi) {
    uint exclMask = EXCL_MASK;
    for (int i=idxLo; i<idxHi; i++) {
        if ((exclusionIds_shr[i] & exclMask) == otherId) {
            return exclusionIds_shr[i] & (~exclMask);
        }
        
    }
    return 0;
}

__device__ int assignFromCell(float3 pos, int idx, uint myId, hipTextureObject_t xs, int *nlistIdxs, int *gridCellArrayIdxs, hipTextureObject_t idToIdxs, int squareIdx, float3 offset, float3 trace, float neighCutSqr, int currentNeighborIdx, hipSurfaceObject_t neighborlist, bool justSorted, uint *exclusionIds_shr, int exclIdxLo_shr, int exclIdxHi_shr) {
    uint idxMin = gridCellArrayIdxs[squareIdx];
    uint idxMax = gridCellArrayIdxs[squareIdx+1];
    for (uint i=idxMin; i<idxMax; i++) {
        int xIdx = XIDX(i, sizeof(float4));
        int yIdx = YIDX(i, sizeof(float4));
        float4 otherPosWhole = tex2D<float4>(xs, xIdx, yIdx); 
        float3 otherPos = make_float3(otherPosWhole);
        float3 distVec = otherPos + (offset * trace) - pos;
        uint otherId = *(uint *) &otherPosWhole.w;

        if (myId != otherId && dot(distVec, distVec) < neighCutSqr/* && !(isExcluded(otherId, exclusions, numExclusions, maxExclusions))*/) {
            uint exclusionTag = addExclusion(otherId, exclusionIds_shr, exclIdxLo_shr, exclIdxHi_shr);
            int xAddrNeigh = XIDX(currentNeighborIdx, sizeof(uint)) * sizeof(uint);
            int yIdxNeigh = YIDX(currentNeighborIdx, sizeof(uint));
            if (justSorted) {
                surf2Dwrite(i | exclusionTag, neighborlist, xAddrNeigh, yIdxNeigh);
            } else {
                int xIdxID = XIDX(otherId, sizeof(int));
                int yIdxID = YIDX(otherId, sizeof(int));
                uint otherIdx = tex2D<int>(idToIdxs, xIdxID, yIdxID);
                surf2Dwrite(otherIdx | exclusionTag, neighborlist, xAddrNeigh, yIdxNeigh);

            }
            currentNeighborIdx ++;
        }

    }
    return currentNeighborIdx;
}
__global__ void assignNeighbors(hipTextureObject_t xs, int nAtoms, hipTextureObject_t idToIdxs, int *nlistIdxs, int *gridCellArrayIdxs, float3 os, float3 ds, int3 ns, float3 periodic, float3 trace, float neighCutSqr, bool justSorted, hipSurfaceObject_t neighborlist, int *exclusionIndexes, uint *exclusionIds, int maxExclusionsPerAtom) {
  ///  extern __shared__ int exclusions_shr[]; 

    extern __shared__ uint exclusionIds_shr[];
    /*
    int tidLo = blockIdx.x * blockDim.x;
    int tidHi = min((blockIdx.x+1) * blockDim.x, nAtoms) - 1;
    int idLo = *(int *) &tex2D<float4>(xs, XIDX(tidLo, sizeof(float4)), YIDX(tidLo, sizeof(float4))).w;
    int idHi = *(int *) &tex2D<float4>(xs, XIDX(tidHi, sizeof(float4)), YIDX(tidHi, sizeof(float4))).w;
    int copyLo = exclusionIndexes[idLo];
    int copyHi = exclusionIndexes[idHi+1];

    copyToShared<uint>(exclusionIds + copyLo, exclusionIds_shr, copyHi - copyLo);
    __syncthreads();
    */
    //so the exclusions that this contiguous block of atoms needs are scattered around the exclusionIndexes list because they're sorted by id.  Need to copy it into shared.  Each thread has to copy from diff block b/c scatted
    int idx = GETIDX();
    float4 posWhole;
    int xIdxAtom, yIdxAtom, myId;
    int exclIdxLo_shr, exclIdxHi_shr, numExclusions;
    exclIdxLo_shr = threadIdx.x * maxExclusionsPerAtom;
    if (idx < nAtoms) {
        xIdxAtom = XIDX(idx, sizeof(float4));
        yIdxAtom = YIDX(idx, sizeof(float4));
        posWhole = tex2D<float4>(xs, xIdxAtom, yIdxAtom);
        myId = *(int *)&posWhole.w;
        int exclIdxLo = exclusionIndexes[myId];
        int exclIdxHi = exclusionIndexes[myId+1];
        numExclusions = exclIdxHi - exclIdxLo;
        exclIdxHi_shr = exclIdxLo_shr + numExclusions;
        for (int i=exclIdxLo; i<exclIdxHi; i++) {
            uint exclusion = exclusionIds[i];
            exclusionIds_shr[maxExclusionsPerAtom*threadIdx.x + i - exclIdxLo] = exclusion;
            //printf("I am thread %d and I am copying %u from global %d to shared %d\n", threadIdx.x, exclusion, i, maxExclusionsPerAtom*threadIdx.x+i-exclIdxLo);
        }
    }
    //okay, now we have exclusions copied into shared
    __syncthreads();
    //YOU JUST NEED TO UPDATE HOW WE CHECK EXCLUSIONS (IDXS IN SHEARED)
    if (idx < nAtoms) {
        //printf("threadid %d idx %x has lo, hi of %d, %d\n", threadIdx.x, idx, exclIdxLo_shr, exclIdxHi_shr);





        float3 pos = make_float3(posWhole);
        int3 sqrIdx = make_int3((pos - os) / ds);
        int xIdx, yIdx, zIdx;
        int xIdxLoop, yIdxLoop, zIdxLoop;
        float3 offset = make_float3(0, 0, 0);
       
        int myIdx;
        if (justSorted) {
            myIdx = idx;
        } else {
            int xIdxID = XIDX(myId, sizeof(int));
            int yIdxID = YIDX(myId, sizeof(int));
            myIdx = tex2D<int>(idToIdxs, xIdxID, yIdxID);
        }

        int currentNeighborIdx = nlistIdxs[myIdx];
        for (xIdx=sqrIdx.x-1; xIdx<=sqrIdx.x+1; xIdx++) {
            offset.x = -floorf((float) xIdx / ns.x);
            xIdxLoop = xIdx + ns.x * offset.x;
            if (periodic.x || (!periodic.x && xIdxLoop == xIdx)) {

                for (yIdx=sqrIdx.y-1; yIdx<=sqrIdx.y+1; yIdx++) {
                    offset.y = -floorf((float) yIdx / ns.y);
                    yIdxLoop = yIdx + ns.y * offset.y;
                    if (periodic.y || (!periodic.y && yIdxLoop == yIdx)) {

                        for (zIdx=sqrIdx.z-1; zIdx<=sqrIdx.z+1; zIdx++) {
                            offset.z = -floorf((float) zIdx / ns.z);
                            zIdxLoop = zIdx + ns.z * offset.z;
                            if (periodic.z || (!periodic.z && zIdxLoop == zIdx)) {
                                int3 sqrIdxOther = make_int3(xIdxLoop, yIdxLoop, zIdxLoop);
                                int sqrIdxOtherLin = LINEARIDX(sqrIdxOther, ns);

                                currentNeighborIdx = assignFromCell(pos, idx, myId, xs, nlistIdxs, gridCellArrayIdxs, idToIdxs, sqrIdxOtherLin, -offset, trace, neighCutSqr, currentNeighborIdx, neighborlist, justSorted, exclusionIds_shr, exclIdxLo_shr, exclIdxHi_shr);

                            }
                        }
                    }
                }


            }
        }
    }
}

void GridGPU::initArrays() {
    perCellArray = GPUArrayDevice<int>(prod(ns) + 1);
    perAtomArray = GPUArrayDevice<int>(state->atoms.size()+1);
    numNeighbors = vector<int>(state->atoms.size()+1, 0);
}

GridGPU::GridGPU(State *state_, float3 ds_, float3 dsOrig_, float3 os_, int3 ns_) : state(state_), ds(ds_), dsOrig(dsOrig_), os(os_), ns(ns_), neighborlist(hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned)){
    initArrays();
};
GridGPU::GridGPU(State *state_, float dx_, float dy_, float dz_) : state(state_), neighborlist(hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned)) {
	Vector trace = state->bounds.trace; //EEHHHHH SHOULD CHANGE TO BOUNDSGPU, but it doesn't really matter because you initialize them at the same time.  FOR NOW
	Vector attemptDDim = Vector(dx_, dy_, dz_);
	VectorInt nGrid = trace / attemptDDim; //so rounding to bigger grid
	Vector actualDDim = trace / nGrid; 
	//making grid that is exactly size of box.  This way can compute offsets easily from Grid that doesn't have to deal with higher-level stuff like bounds	
	is2d = state->is2d;
	ns = nGrid.asInt3();
	ds = actualDDim.asFloat3();
	os = state->boundsGPU.lo;
	if (is2d) {
		ns.z=1;
		ds.z=1;
		assert(os.z==-.5);
	}
	dsOrig = actualDDim.asFloat3();
    initArrays();
};



/*
void printNeighborCounts(int *counts, int nAtoms) {
    cout << "neighbor counts" << endl;
    for (int i=0; i<nAtoms; i++) {
        cout << "n " <<  counts[i+1] - counts[i] << endl;
    }
    cout << "end" << endl;
}
*/
/*
__global__ void printStuff(int *vals, int n) {
    int idx = GETIDX();
    if (idx < n) {
        printf("%d: %d\n", idx, vals[idx]);
    }
}
*/

/*
void __global__ printBiz(hipTextureObject_t tex, int n) {
    int idx = GETIDX();
    if (idx < n) {
        printf("%d is %d\n", idx, tex2D<int>(tex, XIDX(idx), YIDX(idx)));

    }
}


void __global__ printInts(int *xs, int n) {
    int idx = GETIDX();
    if (idx < n) {
        printf("%d is %d\n", idx, xs[idx]);
    }
}
*/
//nAtoms, neighborlist.surf, perAtomArray.ptr, exclusionIndexes.ptr, exclusionIds.ptr, state->gpd.xs.getTex(activeIdx));

/*
void __global__ addExclusions(int nAtoms, hipSurfaceObject_t nlist, int *nlistIdxs, int *exclusionIndexes, uint *exclusionIds, hipTextureObject_t xs) {
    uint exclusionMask = ~(3 << 30);
    extern __shared__ uint exclusionIds_shr[];
    int tidLo = blockIdx.x * blockDim.x;
    int tidHi = min((blockIdx.x+1) * blockDim.x, nAtoms) - 1;
    int idLo = *(int *) &tex2D<float4>(xs, XIDX(tidLo, sizeof(float4), YIDX(tidLo, sizeof(float4)))).w;
    int idHi = *(int *) &tex2D<float4>(xs, XIDX(tidHi, sizeof(float4), YIDX(tidHi, sizeof(float4)))).w;
    int copyLo = exclusionIndexes[idLo];
    int copyHi = exclusionIndexes[idHi+1];

    copyToShared<uint>(exclusionIds + copyLo, exclusionIds_shr, copyHi - copyLo);
    __syncthreads();
    //okay, now all of the exclusions are copied into shared
    int idx = GETIDX();
    if (idx < nAtoms) {
        int id = *(int *) &tex2D<float4>(xs, XIDX(idx, sizeof(float4), YIDX(idx, sizeof(float4)))).w;
        int idxLo_shr = exclusionIndexes[id] - copyLo;
        int idxHi_shr = exclusionIndexes[id+1] - copyLo;
        int nlistIdxLo = nlistIdxs[idx];
        int nlistIdxHi = nlistIdxs[idx+1];
        for (int nlistIdx=nlistIdxLo; nlistIdx<nlistIdxHi; nlistIdx++) {
            uint neighborId = tex2D<uint>(nlist, XIDX(nlistIdx, sizeof(uint)), YIDX(nlistIdx, sizeof(float4)));
            for (int i=idxLo_shr; i<idxHi_shr; i++) {
                uint excl = exclusionIds_shr[i];
                if (neighborId == (excl & exclusionMask)) {
                    uint dist = excl & (~exclusionMask);
                    neighborId |= dist;
                    surf2Dwrite(neighborId, nlist, xAddr, yIdx);
                    break;

                }
            }
        }


    }
        int3 sqrIdx = make_int3((make_float3(tex2D<float4>(xs, xIdx, yIdx)) - os) / ds);
    __device__ void copyToShared (T *src, T *dest, int n) {
}
*/

void GridGPU::periodicBoundaryConditions(float neighCut, bool doSort) {
    //hipDeviceSynchronize();
    //cout << "periodic!" << endl << endl << endl;
    //cout << "max excl is " << maxExclusionsPerAtom << endl;
    /*
    int *exclIdx = exclusionIndexes.get((int *) NULL);
    uint *exclId = exclusionIds.get((uint *) NULL);
    cout << " idxs" << endl;
    for (int i=0; i<exclusionIndexes.n; i++) {
        cout << exclIdx[i] << endl;
    }
    cout << "ids" << endl;
    for (int i=0; i<exclusionIds.n; i++) {
        uint masked = exclId[i] & EXCL_MASK;
        uint dist = exclId[i] >> 30;
        cout << "id " << masked<< endl;
        cout << "dist " << dist << endl;
    }
    */
    float3 ds_orig = ds;
    float3 os_orig = os;
    ds += make_float3(EPSILON, EPSILON, EPSILON); //as defined in Vector.h.  PAIN AND NUMERICAL ERROR AWAIT ALL THOSE WHO ALTER THIS LINE
    os -= make_float3(EPSILON, EPSILON, EPSILON);
    Vector nsV = Vector(make_float3(ns));
    int nAtoms = state->atoms.size();
    BoundsGPU bounds = state->boundsGPU;
    BoundsGPU boundsUnskewed = bounds.unskewed();
    float3 trace = boundsUnskewed.trace();
    int activeIdx = state->gpd.activeIdx;
    if (bounds.sides[0].y or bounds.sides[1].x) {
        Mod::unskewAtoms<<<NBLOCK(nAtoms), PERBLOCK>>>(state->gpd.xs.getSurf(activeIdx), nAtoms, bounds.sides[0], bounds.sides[1], bounds.lo);
    }
    periodicWrap<<<NBLOCK(nAtoms), PERBLOCK>>>(state->gpd.xs.getSurf(), nAtoms, boundsUnskewed);
    int numGridCells = prod(ns);
    if (numGridCells + 1 != perCellArray.n) {
        perCellArray = GPUArrayDevice<int>(numGridCells + 1);
    }
    perCellArray.memset(0);
    perAtomArray.memset(0);
  //  hipDeviceSynchronize();
    countNumInGridCells<<<NBLOCK(nAtoms), PERBLOCK>>>(state->gpd.xs.getTex(), nAtoms, perCellArray.ptr, perAtomArray.ptr, os, ds, ns);
    int *gridCellCounts_h = perCellArray.get((int *) NULL);
    hipDeviceSynchronize();

    
    cumulativeSum(gridCellCounts_h, perCellArray.n);//repurposing this as starting indexes for each grid square

    perCellArray.set(gridCellCounts_h);
    int gridIdx;
    if (doSort) {
        sortPerAtomArrays<<<NBLOCK(nAtoms), PERBLOCK>>>(

                state->gpd.xs.getSurf(activeIdx),  
                state->gpd.xs.getSurf(!activeIdx),

                state->gpd.vs(activeIdx),
                state->gpd.vs(!activeIdx),

                state->gpd.fs(activeIdx),
                state->gpd.fs(!activeIdx),

                state->gpd.fsLast(activeIdx),
                state->gpd.fsLast(!activeIdx),

                state->gpd.types.getSurf(activeIdx),
                state->gpd.types.getSurf(!activeIdx),

                state->gpd.qs.getSurf(activeIdx),
                state->gpd.qs.getSurf(!activeIdx),

                state->gpd.idToIdxs.getSurf(),

                perCellArray.ptr, perAtomArray.ptr, nAtoms, os, ds, ns
                );
        activeIdx = state->gpd.switchIdx();
        gridIdx = activeIdx;
    } else { //otherwise, just use non-active xs array as grid storage
        //gridCPU(state->gpd.xs, activeIdx, nAtoms, perCellArray, perAtomArray, os, ds, ns);
        gridNonSort<<<NBLOCK(nAtoms), PERBLOCK>>>(state->gpd.xs.getTex(activeIdx), state->gpd.xs.getSurf(!activeIdx), nAtoms, perCellArray.ptr, perAtomArray.ptr, os, ds, ns);
        gridIdx = !activeIdx;

    }

    perAtomArray.memset(0);
    countNumNeighbors<<<NBLOCK(nAtoms), PERBLOCK/*, PERBLOCK*sizeof(int)*(state->maxExclusions)*/>>>(state->gpd.xs.getTex(gridIdx), nAtoms, state->gpd.idToIdxs.getTex(), perAtomArray.ptr, perCellArray.ptr, os, ds, ns, bounds.periodic, trace, neighCut*neighCut, doSort);//, state->gpd.nlistExclusionIdxs.getTex(), state->gpd.nlistExclusions.getTex(), state->maxExclusions);
    perAtomArray.get(numNeighbors.data());
    hipDeviceSynchronize();
    
    cumulativeSum(numNeighbors.data(), numNeighbors.size());  //okay, now this is the start index (+1 is end index) of each atom's neighbors
    perAtomArray.set(numNeighbors.data());
    int totalNumNeighbors = numNeighbors.back();
    neighborlist.resize(totalNumNeighbors); //look at method, doesn't always realloc
    assignNeighbors<<<NBLOCK(nAtoms), PERBLOCK, PERBLOCK*maxExclusionsPerAtom*sizeof(uint)>>>(
            state->gpd.xs.getTex(gridIdx), 
            nAtoms, 
            state->gpd.idToIdxs.getTex(), 
            perAtomArray.ptr, 
            perCellArray.ptr, 
            os, ds, ns, bounds.periodic, trace, neighCut*neighCut, doSort, neighborlist.surf,
            exclusionIndexes.ptr, exclusionIds.ptr, maxExclusionsPerAtom
            
            );//, state->gpd.nlistExclusionIdxs.getTex(), state->gpd.nlistExclusions.getTex(), state->maxExclusions);


    //printNeighbors<<<NBLOCK(state->atoms.size()), PERBLOCK>>>(perAtomArray.ptr, neighborlist.tex, state->atoms.size());
    /*
    int *neighCounts = perAtomArray.get((int *) NULL);
    hipDeviceSynchronize();
   printNeighborCounts(neighCounts, state->atoms.size());
   free(neighCounts);
   */
    if (bounds.sides[0].y or bounds.sides[1].x) {
        Mod::skewAtomsFromZero<<<NBLOCK(nAtoms), PERBLOCK>>>(state->gpd.xs.getSurf(activeIdx), nAtoms, bounds.sides[0], bounds.sides[1], bounds.lo);
    }
    free(gridCellCounts_h);
    ds = ds_orig;
    os = os_orig;
 //   verifyNeighborlists(neighCut);

    


}


vector<int> toNeighborCounts(int *idxs, int nAtoms) {
    vector<int> nneigh;
    for (int i=0; i<nAtoms; i++) {
      //  cout << idxs[i] << endl;;
        nneigh.push_back(idxs[i+1]-idxs[i]);
    }
    //cout << idxs[nAtoms] << endl;
    return nneigh;
}


bool GridGPU::verifyNeighborlists(float neighCut) {
    uint *nlist = neighborlist.get((uint *) NULL);
    
    float cutSqr = neighCut * neighCut;
    int *neighIdxs = perAtomArray.get((int *) NULL);
    state->gpd.xs.dataToHost();
    hipDeviceSynchronize();
    vector<int> nneigh = toNeighborCounts(neighIdxs, state->atoms.size());
    vector<float4> xs = state->gpd.xs.h_data;
    state->gpd.xs.dataToHost(!state->gpd.xs.activeIdx);
    hipDeviceSynchronize();
    vector<float4> sortedXs = state->gpd.xs.h_data;
    
  //  int gpuId = *(int *)&sortedXs[TESTIDX].w;

//    int cpuIdx = gpuId;
    vector<vector<int> > cpu_neighbors;
    for (int i=0; i<xs.size(); i++) {
        vector<int> atom_neighbors;
        float3 self = make_float3(xs[i]);
        for (int j=0; j<xs.size(); j++) {
            if (i!=j) {
                float4 otherWhole = xs[j];
                float3 minImage = state->boundsGPU.minImage(self - make_float3(otherWhole));
                if (lengthSqr(minImage) < cutSqr) {
                    atom_neighbors.push_back(*(int*)&otherWhole.w);
                }

            }
        }
        sort(atom_neighbors.begin(), atom_neighbors.end());
        cpu_neighbors.push_back(atom_neighbors);
    }
//    cout << "cpu dist is " << sqrt(lengthSqr(state->boundsGPU.minImage(xs[0]-xs[1])))  << endl;
    for (int i=0; i<xs.size(); i++) {
        vector<int> neighIds;
    //    cout << "begin end " << neighIdxs[i] << " " << neighIdxs[i+1] << endl;
        for (int nIdx=neighIdxs[i]; nIdx<neighIdxs[i+1]; nIdx++) {
      //      cout << "idx " << nlist[nIdx] << endl;
            float4 atom = xs[nlist[nIdx]];
            int id = *(int *) &atom.w;
       //     cout << "id is " << id << endl;
            neighIds.push_back(id);
        }
        sort(neighIds.begin(), neighIds.end());
        if (neighIds != cpu_neighbors[i]) {
            cout << "problem at idx " << i << " id " << *(int *) &xs[i].w << endl;
            cout << "cpu " << cpu_neighbors[i].size() << " gpu " << neighIds.size() << endl;
            for (int x : cpu_neighbors[i]) {
                cout << x << " ";
            }
            cout << endl;
            for (int x : neighIds) {
                cout << x << " ";
            }
            cout << endl;
            break;

        }

    }
    /*
    bool pass = true;
    for (int i=0; i<xs.size(); i++) {
        if (nneigh[i] != cpu_check[i]) {
            vector<int> gpuIdxs, cpuIdxs;
            for (int listIdx=neighIdxs[i]; listIdx < neighIdxs[i+1]; listIdx++) {
                gpuIdxs.push_back(nlist[listIdx]);
            }
            for (int j=0; j<xs.size(); j++) {
                if (i!=j) {
                    float3 minImage = state->boundsGPU.minImage(xs[i] - xs[j]);
                    if (lengthSqr(minImage) < cutSqr) {
                        cpuIdxs.push_back(j);
                    }

                }
            }
            for (int nIdx : gpuIdxs) {
                if (find(cpuIdxs.begin(), cpuIdxs.end(), nIdx) == cpuIdxs.end()) {
                    cout << "cpu is missing neighbor with dist " << length(state->boundsGPU.minImage(xs[i]-xs[nIdx])) << endl;
                    cout << Vector(xs[i]) << "      " << Vector(xs[nIdx]) << "    " << nIdx << endl;
                }
            }
            for (int nIdx : cpuIdxs) {
                if (find(gpuIdxs.begin(), gpuIdxs.end(), nIdx) == gpuIdxs.end()) {
                    cout << "gpu is missing neighbor with dist " << length(state->boundsGPU.minImage(xs[i]-xs[nIdx])) << endl;
                    cout << Vector(xs[i]) << "      " << Vector(xs[nIdx]) << "    " << nIdx << endl;
                }
            }

            cout << nneigh[i] << " on gpu " << cpu_check[i] << " on cpu " << endl;
            //cout << Vector(xs[i]) << endl;
            pass = false;
        }
    }
    if (pass) {
    //    cout << "neighbor count passed" << endl;
    }
    */
    free(nlist);
    free(neighIdxs);
    return true;

}
bool GridGPU::checkSorting(int gridIdx, int *gridIdxs, GPUArrayDevice<int> &gridIdxsDev) {
   // printInts<<<NBLOCK(gridIdxsDev.n), PERBLOCK>>>(gridIdxsDev.ptr, gridIdxsDev.n);
    int numGridIdxs = prod(ns);
    vector<int> activeIds = LISTMAPREF(Atom, int, atom, state->atoms, atom.id);
    vector<int> gpuIds;

    gpuIds.reserve(activeIds.size());
    state->gpd.xs.dataToHost(gridIdx);
    hipDeviceSynchronize();
    vector<float4> &xs = state->gpd.xs.h_data;
    bool correct = true;
    for (int i=0; i<numGridIdxs; i++) {
        int gridLo = gridIdxs[i];
        int gridHi = gridIdxs[i+1];
     //   cout << "hi for " << i << " is " << gridHi << endl;
        for (int atomIdx=gridLo; atomIdx<gridHi; atomIdx++) {
            float4 posWhole = xs[atomIdx];
            float3 pos = make_float3(posWhole);
            int id = *(int *) &posWhole.w;
            gpuIds.push_back(id);
            int3 sqr = make_int3((pos - os) / ds);
            int linear = LINEARIDX(sqr, ns);
            if (linear != i) {
                correct = false;
            }
        }
    }
    sort(activeIds.begin(), activeIds.end());
    sort(gpuIds.begin(), gpuIds.end());
    cout << activeIds.size() << " " << gpuIds.size() << endl;
    if (activeIds != gpuIds) {
        correct = false;
        cout << "different ids!   Seriou problem!" << endl;
        assert(activeIds.size() == gpuIds.size());
    }
    return correct;


    
}


void GridGPU::prepareForRun() {

    const ExclusionList exclList = generateExclusionList(4);
    vector<int> idxs;
    vector<uint> excludedById;
    excludedById.reserve(state->maxIdExisting+1);
    
    auto fillToId = [&] (int id) { //paired list is indexed by id.  Some ids could be missing, so need to fill in empty values
        while (idxs.size() <= id) {
            idxs.push_back(excludedById.size());
        }
    };

    uint exclusionTags[3] = {(uint) 1 << 30, (uint) 2 << 30, (uint) 3 << 30};
    maxExclusionsPerAtom = 0;
    for (auto it = exclList.begin(); it!=exclList.end(); it++) { //is ordered map, so it sorted by ascending id
        int id = it->first;
        //cout << "id is " << id << endl;
        const vector<set<int> > &atomExclusions = it->second;
        fillToId(id); 
        //cout << "filled" << endl;
        //for (int id : idxs) {
        //    cout << id << endl;
        //}
        for (int i=0; i<atomExclusions.size(); i++) {
            const set<int> &idsAtLevel = atomExclusions[i];
            for (auto itId=idsAtLevel.begin(); itId!=idsAtLevel.end(); itId++) {
                uint id = *itId;
                id |= exclusionTags[i];
                excludedById.push_back(id);


            }
        }
        idxs.push_back(excludedById.size());
        maxExclusionsPerAtom = fmax(maxExclusionsPerAtom, idxs.back() - idxs[idxs.size()-2]);
    }
  //  cout << "max excl per atom is " << maxExclusionsPerAtom << endl;
    exclusionIndexes = GPUArrayDevice<int>(idxs.size());
    exclusionIndexes.set(idxs.data());
    exclusionIds = GPUArrayDevice<uint>(excludedById.size());
    exclusionIds.set(excludedById.data());
    //atoms is sorted by id.  list of ids may be sparse, so need to make sure there's enough shared memory for PERBLOCK _atoms_, not just PERBLOCK ids (when calling assign exclusions kernel)

       //for test output
    /*
    cout << "index ptrs " << endl;
    for (int id : idxs) {
        cout << id << endl;
    }
    cout << "end" << endl;
    for (int i=0; i<idxs.size()-1; i++) {
        for (int exclIdx=idxs[i]; exclIdx < idxs[i+1]; exclIdx++) {
            uint excl = excludedById[exclIdx];
            uint filter = (uint) 3 << 30;
            cout << filter << endl;
            uint dist = (excl & filter) >> 30;
            uint id = excl & (~filter);
            cout << "id " << i << " excludes " << id << " with dist " << dist << endl;
        }
    }
    */
}




bool GridGPU::closerThan(const ExclusionList &exclude, 
                       int atomid, int otherid, int16_t depthi) {
    bool closerThan = false;
    // because we want to check lower depths
    --depthi;
    while (depthi >= 0) {
        const set<int> &closer = exclude.at(atomid)[depthi];
        closerThan |= (closer.find(otherid) != closer.end());
        --depthi;
    }
    // atoms are closer to themselves than any other depth away
    closerThan |= (atomid == otherid);
    return closerThan;
}

// allows us to extract any type of Bond from a BondVariant
class bondDowncast : public boost::static_visitor<const Bond &> {
	const BondVariant &_bv;
	public:
		bondDowncast(BondVariant &bv) : _bv(bv) {}
		template <typename T>
		const Bond &operator()(const T &b) const {
			return boost::get<T>(_bv);
		}
};

GridGPU::ExclusionList GridGPU::generateExclusionList(const int16_t maxDepth) {
    
    ExclusionList exclude;
    // not called depth because it's really the depth index, which is one
    // smaller than the depth
    int16_t depthi = 0;
    
    // computes adjacent bonds (depth -> 1, depthi -> 0)
    vector<vector<BondVariant> *> allBonds;
    for (Fix *f : state->fixes) {
        vector<BondVariant> *fixBonds = f->getBonds();
        if (fixBonds != nullptr) {
            allBonds.push_back(fixBonds);
        }
    }
    for (Atom atom : state->atoms) {
        exclude[atom.id].push_back(set<int>());
    }

		//typedef map<int, vector<set<int>>> ExclusionList;
    for (vector<BondVariant> *fixBonds : allBonds) {
        for (BondVariant &bondVariant : *fixBonds) {
			// boost variant magic that takes any BondVariant and turns it into a Bond
            const Bond &bond = boost::apply_visitor(bondDowncast(bondVariant), bondVariant);
            // atoms in the same bond are 1 away from each other
            exclude[bond.getAtomId(0)][depthi].insert(bond.getAtomId(1));
            exclude[bond.getAtomId(1)][depthi].insert(bond.getAtomId(0));
        }
    }
    depthi++;
    
    // compute the rest
    while (depthi < maxDepth) {
        for (Atom atom : state->atoms) {
            // for every atom at the previous depth away
            exclude[atom.id].push_back(set<int>());
            for (int extendFrom : exclude[atom.id][depthi-1]) {
                // extend to all atoms bonded with it
                exclude[atom.id][depthi].insert(
                  exclude[extendFrom][0].begin(), exclude[extendFrom][0].end());
            }
            // remove all atoms that are already excluded to a lower degree
            // TODO: may be a more efficient way
            for (auto it = exclude[atom.id][depthi].begin();
                 it != exclude[atom.id][depthi].end(); /*blank*/ ) {
                if (closerThan(exclude, atom.id, *it, depthi)) {
                   exclude[atom.id][depthi].erase(it++);
                } else {
                    ++it;
                }
            }
        }
        depthi++;
    }
    return exclude;
}

