#include "hip/hip_runtime.h"
#include "DataSetTemperature.h"
#include "cutils_func.h"
#include "boost_for_export.h"
using namespace std;
using namespace boost::python;

DataSetTemperature::DataSetTemperature(uint32_t groupTag_) : DataSet(groupTag_) {
}

void DataSetTemperature::collect(int64_t turn, BoundsGPU &, int nAtoms, float4 *xs, float4 *vs, float4 *fs, float *engs, Virial *virials, hipDeviceProp_t &prop) {
    tempGPU.d_data.memset(0);
    sumVectorSqr3DTagsOverW<float, float4, N_DATA_PER_THREAD> <<<NBLOCK(nAtoms/ (double) N_DATA_PER_THREAD), PERBLOCK, N_DATA_PER_THREAD*PERBLOCK*sizeof(float)>>>(tempGPU.getDevData(), vs, nAtoms, groupTag, fs, prop.warpSize);
    tempGPU.dataToHost();
    turns.push_back(turn);
    turnsPy.append(turn);
}
void DataSetTemperature::appendValues() {
    int n = * (int *) &tempGPU.h_data[1];
    double tempCur = (double) tempGPU.h_data[0] / n / 3.0; 
    vals.push_back(tempCur);
    valsPy.append(tempCur);
    
}

void DataSetTemperature::prepareForRun() {
    tempGPU = GPUArrayGlobal<float>(2);
}

void export_DataSetTemperature() {
    class_<DataSetTemperature, SHARED(DataSetTemperature), bases<DataSet>, boost::noncopyable > ("DataSetTemperature", no_init)
        ;
}
