#include "hip/hip_runtime.h"
#include "FixNVTRescale.h"
#include "cutils_func.h"

__global__ void sumKeInBounds (float *dest, float4 *src, int n, unsigned int groupTag, float4 *fs, BoundsGPU bounds, int warpSize) {
    extern __shared__ float tmp[]; /*should have length of # threads in a block (PERBLOCK) PLUS ONE for counting shared*/
    int potentialIdx = blockDim.x*blockIdx.x + threadIdx.x;
    if (potentialIdx < n) {
        unsigned int atomGroup = * (unsigned int *) &(fs[potentialIdx].w);
        if (atomGroup & groupTag) {
            float4 val = src[blockDim.x*blockIdx.x + threadIdx.x];
            if (bounds.inBounds(make_float3(val))) {
                tmp[threadIdx.x] = lengthSqrOverW( val ) ;
                atomicAdd(dest+1, 1);
            }
        } else {
            tmp[threadIdx.x] = 0;
        }
    } else {
        tmp[threadIdx.x] = 0;
    }
    __syncthreads();
    reduceByN(tmp, blockDim.x, warpSize);
    if (threadIdx.x == 0) {
        atomicAdd(dest, tmp[0]);
    }
}


FixNVTRescale::FixNVTRescale(SHARED(State) state_, string handle_, string groupHandle_, boost::python::list intervals_, boost::python::list temps_, int applyEvery_, SHARED(Bounds) thermoBounds_ ) : Fix(state_, handle_, groupHandle_, NVTRescaleType, applyEvery_), curIdx(0), tempGPU(GPUArrayDeviceGlobal<float>(2)), finished(false) {
    assert(boost::python::len(intervals_) == boost::python::len(temps_)); 
    assert(boost::python::len(intervals_) > 1);
    int len = boost::python::len(intervals_);
    for (int i=0; i<len; i++) {
        boost::python::extract<double> intPy(intervals_[i]);
        boost::python::extract<double> tempPy(temps_[i]);
        if (!intPy.check() or !tempPy.check()) {
            cout << "Invalid value given to fix with handle " << handle << endl;
            assert(intPy.check() and tempPy.check());
        }
        double interval = intPy;
        double temp = tempPy;
        intervals.push_back(interval);
        temps.push_back(temp);
    }
    thermoBounds = thermoBounds_;

   assert(intervals[0] == 0 and intervals.back() == 1); 

}

FixNVTRescale::FixNVTRescale(SHARED(State) state_, string handle_, string groupHandle_, vector<double> intervals_, vector<double> temps_, int applyEvery_, SHARED(Bounds) thermoBounds_) : Fix(state_, handle_, groupHandle_, NVTRescaleType, applyEvery_), curIdx(0), tempGPU(GPUArrayDeviceGlobal<float>(2)), finished(false) {
    assert(intervals.size() == temps.size());
    intervals = intervals_;
    temps = temps_;
    thermoBounds = thermoBounds_;

    forceSingle = false;
}
bool FixNVTRescale::prepareForRun() {
    usingBounds = thermoBounds != SHARED(Bounds) (NULL);
    if (usingBounds) {
        assert(state == thermoBounds->state);
        boundsGPU = thermoBounds->makeGPU();
    }
    return true;
}

void __global__ rescale(int nAtoms, uint groupTag, float4 *vs, float4 *fs, float tempSet, float *tempCurPtr) {
    int idx = GETIDX();
    float2 vals = ((float2 *) tempCurPtr)[0];
    float sumKe = vals.x;
    int n = * (int *) &(vals.y);
    if (vals.x > 0 and idx < nAtoms) {
        float tempCur = sumKe / n / 3.0f; //1th entry is #in group
        uint groupTagAtom = ((uint *) (fs+idx))[3];
        if (groupTag & groupTagAtom) {
            float4 vel = vs[idx];
            float w = vel.w;
            vel *= sqrtf(tempSet / tempCur);
            vel.w = w;
            vs[idx] = vel;
        }
    }
}


void __global__ rescaleInBounds(int nAtoms, uint groupTag, float4 *xs, float4 *vs, float4 *fs, float tempSet, float *tempCurPtr, BoundsGPU bounds) {
    int idx = GETIDX();
    float2 vals = ((float2 *) tempCurPtr)[0];
    float sumKe = vals.x;
    int n = * (int *) &(vals.y);
    if (vals.x > 0 and idx < nAtoms) {
        float tempCur = sumKe / n / 3.0f; //1th entry is #in group
        uint groupTagAtom = ((uint *) (fs+idx))[3];
        if (groupTag & groupTagAtom) {
            float3 x = make_float3(xs[idx]);
            if (bounds.inBounds(x)) {
                float4 vel = vs[idx];
                float w = vel.w;
                vel *= sqrtf(tempSet / tempCur);
                vel.w = w;
                vs[idx] = vel;
            }
        }
    }
}
/*
    template <class K, class T>
__global__ void SUMTESTS (K *dest, T *src, int n, unsigned int groupTag, float4 *fs, int warpSize) {
    extern __shared__ K tmp[];
    int potentialIdx = blockDim.x*blockIdx.x + threadIdx.x;
    if (potentialIdx < n) {
        unsigned int atomGroup = * (unsigned int *) &(fs[potentialIdx].w);
        if (atomGroup & groupTag) {
            tmp[threadIdx.x] = lengthSqrOverW ( src[blockDim.x*blockIdx.x + threadIdx.x])  ;
            atomicAdd(dest+1, 1);
        } else {
            tmp[threadIdx.x] = 0;
        }
    } else {
        tmp[threadIdx.x] = 0;
    }
    __syncthreads();
    int curLookahead = 1;
    int maxLookahead = log2f(blockDim.x-1);
    for (int i=0; i<=maxLookahead; i++) {
        if (! (threadIdx.x % (curLookahead*2))) {
            tmp[threadIdx.x] += tmp[threadIdx.x + curLookahead];
        }
        curLookahead *= 2;
        if (curLookahead >= warpSize) {
            __syncthreads();
        }
    }
    if (threadIdx.x == 0) {
        atomicAdd(dest, tmp[0]);
    }
}
*/


void FixNVTRescale::compute(bool computeVirials) {

    tempGPU.memset(0);
    int nAtoms = state->atoms.size();
    int64_t turn = state->turn;
    double temp;
    if (finished) {
        temp = temps.back();
    } else {
        double frac = (turn-state->runInit) / (double) state->runningFor;
        while (frac > intervals[curIdx+1] and curIdx < intervals.size()-1) {
            curIdx++;
        }
        double tempA = temps[curIdx];
        double tempB = temps[curIdx+1];
        double intA = intervals[curIdx];
        double intB = intervals[curIdx+1];
        double fracThroughInterval = (frac-intA) / (intB-intA);
        temp = tempB*fracThroughInterval + tempA*(1-fracThroughInterval);
    }
    GPUData &gpd = state->gpd;
    int activeIdx = gpd.activeIdx();
    int warpSize = state->devManager.prop.warpSize;
    if (usingBounds) {
        sumKeInBounds<<<NBLOCK(nAtoms), PERBLOCK, PERBLOCK*sizeof(float)>>>(tempGPU.data(), gpd.vs(activeIdx), nAtoms, groupTag, gpd.fs(activeIdx), boundsGPU, warpSize);
        rescaleInBounds<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, groupTag, gpd.xs(activeIdx), gpd.vs(activeIdx), gpd.fs(activeIdx), temp, tempGPU.data(), boundsGPU);
    } else {
        //SUMTESTS<float, float4> <<<NBLOCK(nAtoms), PERBLOCK, PERBLOCK*sizeof(float)>>>(tempGPU.data(), gpd.vs(activeIdx), nAtoms, groupTag, gpd.fs(activeIdx), warpSize);
        sumVectorSqr3DTagsOverW<float, float4> <<<NBLOCK(nAtoms), PERBLOCK, PERBLOCK*sizeof(float)>>>(tempGPU.data(), gpd.vs(activeIdx), nAtoms, groupTag, gpd.fs(activeIdx), warpSize);
        //SAFECALL(sumVectorSqr3DTagsOverW<float, float4> <<<NBLOCK(nAtoms), PERBLOCK, PERBLOCK*sizeof(float)>>>(tempGPU.data(), gpd.vs(activeIdx), nAtoms, groupTag, gpd.fs(activeIdx)));
        rescale<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, groupTag, gpd.vs(activeIdx), gpd.fs(activeIdx), temp, tempGPU.data());
        //SAFECALL(rescale<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms, groupTag, gpd.vs(activeIdx), gpd.fs(activeIdx), temp, tempGPU.data()));
    }
}



bool FixNVTRescale::downloadFromRun() {
    finished = true;
    return true;
}


void export_FixNVTRescale() {
    boost::python::class_<FixNVTRescale,
                          SHARED(FixNVTRescale),
                          boost::python::bases<Fix> > (
        "FixNVTRescale",
        boost::python::init<SHARED(State), string, string, boost::python::list,
                            boost::python::list,
                            boost::python::optional<int, SHARED(Bounds)> > (
            boost::python::args("state", "handle", "groupHandle", "intervals",
                                "temps", "applyEvery", "thermoBounds")
        )
    )
    .def_readwrite("finished", &FixNVTRescale::finished)
    .def_readwrite("thermoBounds", &FixNVTRescale::thermoBounds);
    ;
}
