#include "hip/hip_runtime.h"
#include "FixE3B3.h"
#include "BoundsGPU.h"
#include "GridGPU.h"
#include "State.h"
#include "boost_for_export.h"
#include "cutils_math.h"
#include "list_macro.h"

const std::string E3B3Type = "E3B3";
namespace py = boost::python;
/* Constructor
 * Makes an instance of the E3B3 fix
 */

FixE3B3::FixE3B3(boost::shared_ptr<State> state_,
                  std::string handle_,
                  std::string groupHandle_): Fix(state_, handle_, groupHandle_, E3B3Type, true, true, false, 1) { 
    // set the cutoffs used in this potential
    rf = 5.2; // far cutoff for threebody interactions (Angstroms)
    rs = 5.0; // short cutoff for threebody interactions (Angstroms)
    rc = 7.2; // cutoff for our local neighborlist (Angstroms)
    padding = 2.0; // implied since rc - rf = 2.0; pass this to local GridGPU on instantiation
    // to do: set up the local gridGPU for this set of GPUData; 
    // ---- which means we need to set up the local GPUData;
    // ------- can't do this until we have all the atoms in simulation; so do it in prepareForRun
};

//
// what arguments do we need here? we are updating the molecule positions from 
// the current atom positions
// __global__ void compute_COM(int4 *waterIds, float4 *xs, float4 *vs, int *idToIdxs, int nMols, float4 *com, BoundsGPU bounds) {

// from FixRigid.cu
__device__ inline float3 positionsToCOM(float3 *pos, float *mass, float ims) {
  return (pos[0]*mass[0] + pos[1]*mass[1] + pos[2]*mass[2] + pos[3]*mass[3])*ims;
}


// see FixRigid.cu! does the same thing. but now, we store it in their own gpd..
 __global__ void update_xs(int nMolecules, int4 *waterIds, float4 *mol_xs,
                           float4 *xs, float4 *vs, int *idToIdxs, BoundsGPU bounds) {

     // now do pretty much the same as FixRigid computeCOM()
     // --- remember to account for the M-site
    int idx = GETIDX();
    
    if (idx < nMolecules) {

        // may as well make these arrays
        int theseIds[4]; 
        float3 pos[4];
        float mass[4];

        theseIds[0] = waterIds[idx].x;
        theseIds[1] = waterIds[idx].y;
        theseIds[2] = waterIds[idx].z;
        theseIds[3] = waterIds[idx].w;

        float ims = 0.0f;
        for (int i = 0; i < 4; i++) {
            int thisId = theseIds[i];
            int thisIdx = idToIdxs[thisId];
            float3 p = make_float3(xs[thisIdx]);
            pos[i] = p;
            ims += vs[thisIdx].w;
            mass[i] = 1.0f / vs[thisIdx].w;
        }

        for (int i = 1; i < 4; i++) {
            float3 delta = pos[i] - pos[0];
            delta = bounds.minImage(delta);
            pos[i] = pos[0] + delta;
        }

        xs[idx]  = make_float4(positionsToCOM(pos, mass,ims));
        xs[idx].w = ims;
    }

}

void FixE3B3::compute(bool computeVirials) {
    
    // send the molecules to the e3b3 evaluator, where we compute both the two-body correction
    // and the three-body interactions.
    // -- send the correct neighbor list (specific to this potential) and the array of water molecules
    //    local to this gpu
    // -- still need to send the global simulation data, which contains the atoms itself
    
    // get the activeIdx for our local gpd (the molecule-by-molecule stuff);
    int activeIdx = gpd.activeIdx();

    // and the global gpd
    // --- IMPORTANT: the virials must be taken from the /global/ gpudata!
    GPUData &gpdGlobal = state->gpd;
    int globalActiveIdx = gpdGlobal.activeIdx();
    
    // our grid data holding our molecule-by-molecule neighbor list
    // -- we need to copy over the molecule array as well.
    
    // although it says 'perAtomArray', note that all of this gpd for this grid is by molecule
    // so, its just a misnomer in this instance. its a count of neighboring molecules.
    uint16_t *neighborCounts = gridGPU.perAtomArray.d_data.data();

    /* data required for compute_e3b3:
       - nMolecules
       - moleculesIdsToIdxs
       - waterIds (atom IDS in a given molecule)
       - molecules neighborcounts
       - molecules nlist
       - molecules - cumulSumMaxPerBlock (grid.perBlockArray.d_data.data())a
       - warpsize
       - atom idsToIdxs
       - atom positions
       - boundsGPU (state)
       - virials (global)
    */

    if (computeVirials) {
        compute_E3B3<EvaluatorE3B3, true> <<<NBLOCK(nMolecules), PERBLOCK>>> (
            nMolecules, 
            gpd.idToIdxs.d_data.data(), 
            waterIdsGPU.data(),
            gridGPU.perAtomArray.d_data.data(),
            gridGPU.neighborlist.data(), 
            gridGPU.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize,
            gpdGlobal.idToIdxs.d_data.data(), 
            gpdGlobal.xs(globalActiveIdx), 
            gpdGlobal.fs(globalActiveIdx),
            state->boundsGPU, 
            gpdGlobal.virials.d_data.data(),
            evaluator);
        /*
        (int nMolecules, 
         const int *__restrict__ molIdToIdxs,
         const float4 *__restrict__ atomsFromMolecule,
         const uint16_t *__restrict__ neighborCounts, 
         const uint *__restrict__ neighborlist, 
         const uint32_t * __restrict__ cumulSumMaxPerBlock, 
         int warpSize, 
         const int *__restrict__ idToIdxs,
         const float4 *__restrict__ xs, 
         float4 *__restrict__ fs, 
         BoundsGPU bounds, 
         Virial *__restrict__ virials,
         EVALUATOR eval)
        */
    } else {
        compute_E3B3<EvaluatorE3B3, false> <<<NBLOCK(nMolecules), PERBLOCK>>> (
            nMolecules, 
            gpd.idToIdxs.d_data.data(), 
            waterIdsGPU.data(),
            neighborCounts, 
            gridGPU.neighborlist.data(), 
            gridGPU.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize,
            gpdGlobal.idToIdxs.d_data.data(), 
            gpdGlobal.xs(globalActiveIdx), 
            gpdGlobal.fs(globalActiveIdx),
            state->boundsGPU, 
            gpdGlobal.virials.d_data.data(),
            evaluator);
    };
            /*
    evalWrap->compute(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx),
                      neighborCounts, gridGPU.neighborlist.data(), gridGPU.perBlockArray.d_data.data(),
                      state->devManager.prop.warpSize, numTypes, state->boundsGPU,
                      gpd.virials.d_data.data(), computeVirials);
    */
    
    // we will pass the following to our compute function:
    
}

bool FixE3B3::stepInit(){
    // we use this as an opportunity to re-create the local neighbor list, if necessary
    int periodicInterval = state->periodicInterval;
    if (state->turn % periodicInterval == 0) {
        // do the re-creation of the neighborlist for E3B3
        // -- the xs of the molecules is /not/ updated with the atoms!
        //    but this is what we form our neighborlist off of (for the molecule-by-molecule neighborlist)
        //    so, do a kernel call here to update them to the current positions
        //    of their constituent atoms

        // for each thread, we have one molecule
        // -- get the atoms for this idx, compute COM, set the xs to the new value, and return
        //    -- need idToIdx for atoms? I think so.  Also, this is easy place to check 
        //       accessing the data arrays
        uint activeIdx = gpd.activeIdx();

        // get the global gpd and the bounds
        uint globalActiveIdx = state->gpd.activeIdx();
        GPUData &gpdGlobal = state->gpd;
        BoundsGPU &bounds = state->boundsGPU;

        // pass the local gpd (molecule by molecule) and the global (atom by atom) gpd
        // -- -with this, our local gpd data for the molecule COM is up to date with 
        //     the current atomic data
        update_xs<<<NBLOCK(nMolecules), PERBLOCK>>>(
            nMolecules, waterIdsGPU.data(), gpd.xs(activeIdx), 
            gpdGlobal.xs(globalActiveIdx), gpdGlobal.vs(globalActiveIdx), gpdGlobal.idToIdxs.d_data.data(),
            bounds
        );

        
        // our grid now operates on the updated molecule xs to get a molecule by molecule neighborlist    
        gridGPU.periodicBoundaryConditions();
    }
    return true;
}


/* Single Point Eng
   */
void FixE3B3::singlePointEng(float *perParticleEng) {
    // and, the three body contribution
    // -- we still pass everything molecule by molecule... but add it to their particle arrays

    // gonna need to look up how this is done..

}



/* prepareForRun

   */
bool FixE3B3::prepareForRun(){
    
    // units for distance are always angstroms, 
    // in the context of simulations that would use E3B3
    float rs = 5.0;
    float rf = 5.2;

    /* TODO: put in the real values for these parameters */
    // the values for our E3B3 parameters.  should we call state->units here?
    // ---- probably for the Ea, Eb, Ec, E2 constants! But, distance is always in angstroms.
    float E2 = 1.0000;
    float Ea = 1.0000;
    float Eb = 1.0000;
    float Ec = 1.0000;
    float k2 = 1.0000;
    float k3 = 1.0000;
    
    // instantiate the evaluator
    evaluator = EvaluatorE3B3(rs, rf, E2,
                              Ea, Eb, Ec,
                              k2, k3);
    
    // set up the int4 waterMoleculeIds
    int nMolecules = waterMolecules.size();

    waterIdsGPU = GPUArrayDeviceGlobal<int4>(nMolecules);
    waterIdsGPU.set(waterIds.data()); // waterIds vector populated as molecs added
    
    /*
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    */

    // see State.cpp: State::prepareForRun(); most of this code is taken from there;
    // but with molecules now!
    nMolecules = waterMolecules.size();
    
    std::vector<float4> xs_vec;
    std::vector<uint> ids;

    xs_vec.reserve(nMolecules);
    ids.reserve(nMolecules);

    for (auto &molecule: waterMolecules)  {
        Vector this_xs = molecule.COM();
        float4 new_xs = make_float4(this_xs[0], this_xs[1], this_xs[2], 0);
        xs_vec.push_back(new_xs);

        ids.push_back(molecule.id);
    }

    // note that gpd is the /local/ gpd
    gpd.xs.set(xs_vec);
    gpd.ids.set(ids);
   

    std::vector<int> id_vec = LISTMAPREF(Molecule, int, m, waterMolecules, m.id);
    std::vector<int> idToIdxs_vec;
    int size = *std::max_element(id_vec.begin(), id_vec.end()) + 1;
    idToIdxs_vec.reserve(size);
    for (int i=0; i<size; i++) {
        idToIdxs_vec.push_back(-1);
    }
    for (int i=0; i<id_vec.size(); i++) {
        idToIdxs_vec[id_vec[i]] = i;
    }

    gpd.idToIdxsOnCopy = idToIdxs_vec;
    gpd.idToIdxs.set(idToIdxs_vec);
    //gridGPU = grid.makeGPU(maxRCut);  // uses os, ns, ds, dsOrig from AtomGrid
    double maxRCut = rf;// cutoff of our potential (5.2 A)
    double padding = 2.0;
    double gridDim = maxRCut + padding;

    // make a pointer to this gpudata, which we will pass to the local grid
    GPUData *gpuData = &gpd;

    // this number has no meaning whatsoever; it is completely arbitrary;
    // -- we are not using exclusionMode for this grid or set of GPUData
    int exclusionMode = 30;
    // I think this is doubly irrelevant, since we use a doExclusions(false) method later (below)

    gridGPU = GridGPU(state, gridDim, gridDim, gridDim, gridDim, exclusionMode, padding, gpuData);

    // tell gridGPU that the only GPUData we need to sort are positions (and, of course, the molecule/atom id's)
    gridGPU.onlyPositions(true);

    // tell gridGPU not to do any exclusions stuff
    gridGPU.doExclusions(false);

    // so, the only buffers that we need are the xs and ids!
    gpd.xsBuffer = GPUArrayGlobal<float4>(nMolecules);
    //gpd.vsBuffer = GPUArrayGlobal<float4>(nMolecules);
    //gpd.fsBuffer = GPUArrayGlobal<float4>(nMolecules);
    gpd.idsBuffer = GPUArrayGlobal<uint>(nMolecules);
    
    return true;
}


/* restart chunk?


   */



/* postRun
   * nothing to do hereeeee

   */


// the atom ids are presented as the input; assembled into a molecule
void FixE3B3::addMolecule(int id_O, int id_H1, int id_H2, int id_M) {
    
    // id's are arranged as O, H, H, M
    std::vector<int> localWaterIds;

    
    // add to waterIds vector the four atom ids
    localWaterIds.push_back(id_O);
    localWaterIds.push_back(id_H1);
    localWaterIds.push_back(id_H2);
    localWaterIds.push_back(id_M);


    // mass of O > mass H1 == mass H2 > mass M
    bool ordered = true;
    double massO = state->idToAtom(id_O).mass; 
    double massH1 = state->idToAtom(id_H1).mass;
    double massH2 = state->idToAtom(id_H2).mass;
    double massM = state->idToAtom(id_M).mass;

    // check the ordering
    if (! (massO > massH1 && massO > massH2 )) {
        ordered = false;
    }
    if (massH1 != massH2) ordered = false;
    if (!(massH1 > massM)) ordered = false;

    if (! (ordered)) mdError("Ids in FixE3B3::addMolecule must be as O, H1, H2, M");

    // assemble them in to a molecule
    Molecule thisWater = Molecule(state, localWaterIds);

    // append this molecule to the class variable waterMolecules
    // -- molecule id is implicit as the index in this list
    waterMolecules.push_back(thisWater);

    int4 idsAsInt4 = make_int4(localWaterIds[0], localWaterIds[1], localWaterIds[2], localWaterIds[3]);
    // and add to the global list
    waterIds.push_back(idsAsInt4);


}

/* exports

   */

void export_FixE3B3() {
  py::class_<FixE3B3, boost::shared_ptr<FixE3B3>, py::bases<Fix> > ( 
								      "FixE3B3",
								      py::init<boost::shared_ptr<State>, std::string, std::string>
								      (py::args("state", "handle", "groupHandle")
								       ))
    .def("addMolecule", &FixE3B3::addMolecule,
	     (py::arg("id_O"), 
          py::arg("id_H1"), 
          py::arg("id_H2"),
          py::arg("id_M")
         )
	 );
}
