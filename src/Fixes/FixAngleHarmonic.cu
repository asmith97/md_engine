#include "hip/hip_runtime.h"

#include "FixHelpers.h"
#include "helpers.h"
#include "FixAngleHarmonic.h"
#include "cutils_func.h"
#define SMALL 0.0001f
namespace py = boost::python;
__global__ void compute_cu(int nAtoms, float4 *xs, float4 *forces, hipTextureObject_t idToIdxs, AngleGPU *angles, int *startstops, BoundsGPU bounds, AngleHarmonicType *parameters, int nTypes) {
    printf("hey");
    int idx = GETIDX();
    extern __shared__ int all_shr[];
    int idxBeginCopy = startstops[blockDim.x*blockIdx.x];
    int idxEndCopy = startstops[min(nAtoms, blockDim.x*(blockIdx.x+1))];
    AngleGPU *angles_shr = (AngleGPU *) all_shr;
    AngleHarmonicType *parameters_shr = (AngleHarmonicType *) (angles_shr + (idxEndCopy - idxBeginCopy));
    copyToShared<AngleGPU>(angles + idxBeginCopy, angles_shr, idxEndCopy - idxBeginCopy);
    copyToShared<AngleHarmonicType>(parameters, parameters_shr, nTypes);
    __syncthreads();
    if (idx < nAtoms) {
  //      printf("going to compute %d\n", idx);
        int startIdx = startstops[idx];
        int endIdx = startstops[idx+1];
        //so start/end is the index within the entire bond list.
        //startIdx - idxBeginCopy gives my index in shared memory
        int shr_idx = startIdx - idxBeginCopy;
        int n = endIdx - startIdx;
        if (n>0) {
            int myIdxInAngle = angles_shr[shr_idx].type >> 29;
            int idSelf = angles_shr[shr_idx].ids[myIdxInAngle];

            int idxSelf = tex2D<int>(idToIdxs, XIDX(idSelf, sizeof(int)), YIDX(idSelf, sizeof(int)));
            float3 pos = make_float3(xs[idxSelf]);
            //float3 pos = make_float3(float4FromIndex(xs, idxSelf));
            float3 forceSum = make_float3(0, 0, 0);
            for (int i=0; i<n; i++) {
             //   printf("ANGLE! %d\n", i);
                AngleGPU angle = angles_shr[shr_idx + i];
                uint32_t typeFull = angle.type;
                myIdxInAngle = typeFull >> 29;
                int type = static_cast<int>((typeFull << 3) >> 3);
                AngleHarmonicType angleType = parameters_shr[type];
                float3 positions[3];
                positions[myIdxInAngle] = pos;
                int toGet[2];
                if (myIdxInAngle==0) {
                    toGet[0] = 1;
                    toGet[1] = 2;
                } else if (myIdxInAngle==1) {
                    toGet[0] = 0;
                    toGet[1] = 2;
                } else if (myIdxInAngle==2) {
                    toGet[0] = 0;
                    toGet[1] = 1;
                }
                for (int i=0; i<2; i++) {
                    positions[toGet[i]] = make_float3(perAtomFromId(idToIdxs, xs, angle.ids[toGet[i]]));
                }
                for (int i=1; i<3; i++) {
                    positions[i] = positions[0] + bounds.minImage(positions[i]-positions[0]);
                }
                float3 directors[2];
                directors[0] = positions[0] - positions[1];
                directors[1] = positions[2] - positions[1];
             //   printf("position Xs %f %f %f\n", positions[0].x, positions[1].x, positions[2].x);
              //  printf("director Xs %f %f\n", directors[0].x, directors[1].x);
                float distSqrs[2];
                float dists[2];
                for (int i=0; i<2; i++) {
                    distSqrs[i] = lengthSqr(directors[i]);
                    dists[i] = sqrtf(distSqrs[i]);
                }
                float c = dot(directors[0], directors[1]);
             //   printf("prenorm c is %f\n", c);
                float invDistProd = 1.0f / (dists[0]*dists[1]);
              //  printf("inv dist is %f\n", invDistProd);
                c *= invDistProd;
              //  printf("c is %f\n", c);
                if (c>1) {
                    c=1;
                } else if (c<-1) {
                    c=-1;
                }
                float s = sqrtf(1-c*c);
                if (s < SMALL) {
                    s = SMALL;
                }
                s = 1.0f / s;
                float dTheta = acosf(c) - angleType.thetaEq;
             //   printf("current %f theta eq %f idx %d, type %d\n", acosf(c), angleType.thetaEq, myIdxInAngle, type);

                float forceConst = angleType.k * dTheta;
                float a = -2.0f * forceConst * s;
                float a11 = a*c/distSqrs[0];
                float a12 = -a*invDistProd;
                float a22 = a*c/distSqrs[1];
             //   printf("forceConst %f a %f s %f dists %f %f %f\n", forceConst, a, s, a11, a12, a22);

                if (myIdxInAngle==0) {
                    forceSum += ((directors[0] * a11) + (directors[1] * a12)) * 0.5;
                } else if (myIdxInAngle==1) {
                    forceSum -= ((directors[0] * a11) + (directors[1] * a12) + (directors[1] * a22) + (directors[0] * a12)) * 0.5; 
                } else {
                    forceSum += ((directors[1] * a22) + (directors[0] * a12)) * 0.5;
                }
             //   printf("%f %f %f\n", forceSum.x, forceSum.y, forceSum.z);
            }
            float4 curForce = forces[idxSelf];
         //   printf("Final force is %f %f %f\n", forceSum.x, forceSum.y, forceSum.z);
            curForce += forceSum;
            forces[idxSelf] = curForce;
        }
    }
}


FixAngleHarmonic::FixAngleHarmonic(SHARED(State) state_, string handle) : FixPotentialMultiAtom(state_, handle, angleHarmType), pyListInterface(&forcers, &pyForcers) {
    forceSingle = true;
}


void FixAngleHarmonic::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    int activeIdx = state->gpd.activeIdx();
    /*
    for (AngleVariant &aVar : forcers) {
        AngleHarmonic &a = get<AngleHarmonic>(aVar);
        printf("Angle ids k theta %d %d %d %f %f\n", a.ids[0], a.ids[1], a.ids[2], a.k, a.thetaEq);
    }
    */
    cout << "hello" << endl;
    compute_cu<<<NBLOCK(nAtoms), PERBLOCK, sizeof(AngleGPU) * maxForcersPerBlock + parameters.size() * sizeof(AngleHarmonicType)>>>(nAtoms, state->gpd.xs(activeIdx), state->gpd.fs(activeIdx), state->gpd.idToIdxs.getTex(), forcersGPU.data(), forcerIdxs.data(), state->boundsGPU, parameters.data(), parameters.size());

}

//void cumulativeSum(int *data, int n);
//okay, so the net result of this function is that two arrays (items, idxs of items) are on the gpu and we know how many bonds are in bondiest  block

void FixAngleHarmonic::setAngleTypeCoefs(int type, double k, double thetaEq) {
    //cout << type << " " << k << " " << thetaEq << endl;
    assert(thetaEq>=0);
    AngleHarmonic dummy(k, thetaEq);
    setForcerType(type, dummy);
}

void FixAngleHarmonic::createAngle(Atom *a, Atom *b, Atom *c, double k, double thetaEq, int type) {
    vector<Atom *> atoms = {a, b, c};
    validAtoms(atoms);
    if (type == -1) {
        assert(k!=COEF_DEFAULT and thetaEq!=COEF_DEFAULT);
    }
    forcers.push_back(AngleHarmonic(a, b, c, k, thetaEq, type));
    pyListInterface.updateAppendedMember();
}
string FixAngleHarmonic::restartChunk(string format) {
    stringstream ss;

    return ss.str();
}

void export_FixAngleHarmonic() {
    boost::python::class_<FixAngleHarmonic,
                          SHARED(FixAngleHarmonic),
                          boost::python::bases<Fix, TypedItemHolder> > (
        "FixAngleHarmonic",
        boost::python::init<SHARED(State), string> (
                                        boost::python::args("state", "handle"))
    )
    .def("createAngle", &FixAngleHarmonic::createAngle,
            (boost::python::arg("k")=COEF_DEFAULT,
             boost::python::arg("thetaEq")=COEF_DEFAULT,
             boost::python::arg("type")=-1)
        )
    .def("setAngleTypeCoefs", &FixAngleHarmonic::setAngleTypeCoefs,
            (boost::python::arg("type")=-1,
             boost::python::arg("k")=COEF_DEFAULT,
             boost::python::arg("thetaEq")=COEF_DEFAULT
            )
        )
    .def_readonly("angles", &FixAngleHarmonic::pyForcers)
    ;

}

