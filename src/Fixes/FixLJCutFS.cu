#include "hip/hip_runtime.h"
#include "FixLJCutFS.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "PairEvaluateIso.h"
#include "State.h"
#include "cutils_func.h"

const std::string LJCutType = "LJCutFS";

FixLJCutFS::FixLJCutFS(SHARED(State) state_, std::string handle_)
    : FixPair(state_, handle_, "all", LJCutType, true, 1),
      epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut") {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    initializeParameters("FCutHandle", FCuts);
    paramOrder = {epsHandle, sigHandle, rCutHandle,"FCutHandle"};
}
void FixLJCutFS::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

        compute_force_iso<EvaluatorLJFS, 4>  <<<NBLOCK(nAtoms), PERBLOCK, 4*numTypes*numTypes*sizeof(float)>>>(nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx), neig\
hborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU\
, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator);



}

void FixLJCutFS::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

    compute_energy_iso<EvaluatorLJFS, 4><<<NBLOCK(nAtoms), PERBLOCK, 4*numTypes*numTypes*sizeof(float)>>>(nAtoms, gpd.xs(activeIdx), perParticleEng, neighbor\
Counts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, ne\
ighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator);



}

bool FixLJCutFS::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillEps = [] (float a, float b) {
        return sqrt(a*b);
    };

    auto fillSig = [] (float a, float b) {
        return (a+b) / 2.0;
    };
    auto fillRCut = [this] (float a, float b) {
        return (float) std::fmax(a, b);
    };
    auto none = [] (float a){};

    auto fillRCutDiag = [this] () {
        return (float) state->rCut;//WHY??
    };

    auto processEps = [] (float a) {
        return 24*a;
    };
    auto processSig = [] (float a) {
        return pow(a, 6);
    };
    auto processRCut = [] (float a) {
        return a*a;
    };
    
    auto fillFCut = [this] (int a, int b) {
        int numTypes = state->atomParams.numTypes;
        float epstimes24=squareVectorRef<float>(paramMap[epsHandle]->data(),numTypes,a,b);
        float rCutSqr = squareVectorRef<float>(paramMap[rCutHandle]->data(),numTypes,a,b);
        float sig6 = squareVectorRef<float>(paramMap[sigHandle]->data(),numTypes,a,b);
        float p1 = epstimes24*2*sig6*sig6;
        float p2 = epstimes24*sig6;
        float r2inv = 1/rCutSqr;
        float r6inv = r2inv*r2inv*r2inv;
        float forceScalar = r6inv * r2inv * (p1 * r6inv - p2)*sqrt(rCutSqr);

        return forceScalar;
    };
    prepareParameters(epsHandle, fillEps, processEps, false);
    prepareParameters(sigHandle, fillSig, processSig, false);
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);
    prepareParameters("FCutHandle", fillFCut);
    sendAllToDevice();
    return true;
}

std::string FixLJCutFS::restartChunk(std::string format) {
    std::stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}

bool FixLJCutFS::readFromRestart(pugi::xml_node restData) {
    std::cout << "Reading form restart" << std::endl;
    auto curr_param = restData.first_child();
    while (curr_param) {
        if (curr_param.name() == "parameter") {
           std::vector<float> val;
           std::string paramHandle = curr_param.attribute("handle").value();
           std::string s;
           std::istringstream ss(curr_param.value());
           while (ss >> s) {
               val.push_back(atof(s.c_str()));
           }
           initializeParameters(paramHandle, val);
        }
        curr_param = curr_param.next_sibling();
    }
    std::cout << "Reading LJ parameters from restart\n";
    return true;
}

bool FixLJCutFS::postRun() {
    resetToPreproc(sigHandle);
    resetToPreproc(epsHandle);
    resetToPreproc(rCutHandle);

    return true;
}

void FixLJCutFS::addSpecies(std::string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    initializeParameters(rCutHandle, FCuts);

}

std::vector<float> FixLJCutFS::getRCuts() { //to be called after prepare.  These are squares now
    return LISTMAP(float, float, rc, rCuts, sqrt(rc));
}

void export_FixLJCutFS() {
    boost::python::class_<FixLJCutFS,
                          SHARED(FixLJCutFS),
                          boost::python::bases<FixPair>, boost::noncopyable > (
        "FixLJCutFS",
        boost::python::init<SHARED(State), std::string> (
            boost::python::args("state", "handle"))
    );

}