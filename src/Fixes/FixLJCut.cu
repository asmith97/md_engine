#include "hip/hip_runtime.h"
#include "FixLJCut.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "PairEvaluateIso.h"
#include "State.h"
#include "cutils_func.h"
using namespace std;
namespace py = boost::python;
const string LJCutType = "LJCut";

__device__ void eval (float3 &forceSum, float3 dr, float *params, float lenSqr, float multiplier) {
    printf("here\n");
    float rCutSqr = params[2];
    if (lenSqr < rCutSqr) {
        float epstimes24 = params[0];
        float sig6 = params[1];
        float p1 = epstimes24*2*sig6*sig6;
        float p2 = epstimes24*sig6;
        float r2inv = 1/lenSqr;
        float r6inv = r2inv*r2inv*r2inv;
        float forceScalar = r6inv * r2inv * (p1 * r6inv - p2) * multiplier;

        float3 forceVec = dr * forceScalar;
        forceSum += forceVec;
    }
}


FixLJCut::FixLJCut(boost::shared_ptr<State> state_, string handle_)
  : FixPair(state_, handle_, "all", LJCutType, true, 1),
    epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut")
{
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    paramOrder = {epsHandle, sigHandle, rCutHandle};
}

void FixLJCut::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

   SAFECALL(( compute_force_iso<EvaluatorLJ, 3> <<<NBLOCK(nAtoms), PERBLOCK, 3*numTypes*numTypes*sizeof(float)>>>(
            nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx),
            neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
            neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator)
            ));

}

void FixLJCut::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

    compute_energy_iso<EvaluatorLJ, 3><<<NBLOCK(nAtoms), PERBLOCK, 3*numTypes*numTypes*sizeof(float)>>>(nAtoms, gpd.xs(activeIdx), perParticleEng, 
                                                                                                        neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator);



}

bool FixLJCut::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillEps = [] (float a, float b) {
        return sqrt(a*b);
    };

    auto fillSig = [] (float a, float b) {
        return (a+b) / 2.0;
    };
    auto fillRCut = [this] (float a, float b) {
        return (float) std::fmax(a, b);
    };
    auto none = [] (float a){};

    auto fillRCutDiag = [this] () {
        return (float) state->rCut;
    };

    auto processEps = [] (float a) {
        return 24*a;
    };
    auto processSig = [] (float a) {
        return pow(a, 6);
    };
    auto processRCut = [] (float a) {
        return a*a;
    };
    prepareParameters(epsHandle, fillEps, processEps, false);
    prepareParameters(sigHandle, fillSig, processSig, false);
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);
    sendAllToDevice();
    return true;
}

string FixLJCut::restartChunk(string format) {
    stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}

bool FixLJCut::readFromRestart(pugi::xml_node restData) {
    cout << "Reading form restart" << endl;
    auto curr_param = restData.first_child();
    while (curr_param) {
        if (curr_param.name() == "parameter") {
            vector<float> val;
            string paramHandle = curr_param.attribute("handle").value();
            string s;
            istringstream ss(curr_param.value());
            while (ss >> s) {
                val.push_back(atof(s.c_str()));
            }
            initializeParameters(paramHandle, val);
        }
        curr_param = curr_param.next_sibling();
    }
    cout << "Reading LJ parameters from restart" << endl;
    return true;
}

bool FixLJCut::postRun() {

    return true;
}

void FixLJCut::addSpecies(string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);

}

vector<float> FixLJCut::getRCuts() { 
    vector<float> res;
    vector<float> &src = *(paramMap[rCutHandle]);
    for (float x : src) {
        if (x == DEFAULT_FILL) {
            res.push_back(-1);
        } else {
            res.push_back(x);
        }
    }

    return res;
}

void export_FixLJCut() {
    py::class_<FixLJCut, boost::shared_ptr<FixLJCut>, py::bases<FixPair>, boost::noncopyable > (
        "FixLJCut",
        py::init<boost::shared_ptr<State>, string> (py::args("state", "handle"))
    );

}
