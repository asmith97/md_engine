#include "hip/hip_runtime.h"
#include "FixLJCut.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "PairEvaluateIso.h"
#include "State.h"
#include "cutils_func.h"
using namespace std;
namespace py = boost::python;
const string LJCutType = "LJCut";



FixLJCut::FixLJCut(boost::shared_ptr<State> state_, string handle_)
  : FixPair(state_, handle_, "all", LJCutType, true, false, 1),
    epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut")
{
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    paramOrder = {rCutHandle, epsHandle, sigHandle};
}

void FixLJCut::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

   SAFECALL(( compute_force_iso<EvaluatorLJ, 3> <<<NBLOCK(nAtoms), PERBLOCK, 3*numTypes*numTypes*sizeof(float)>>>(
            nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx),
            neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
            neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator)
            ));

}

void FixLJCut::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

    compute_energy_iso<EvaluatorLJ, 3><<<NBLOCK(nAtoms), PERBLOCK, 3*numTypes*numTypes*sizeof(float)>>>(nAtoms, gpd.xs(activeIdx), perParticleEng, 
                                                                                                        neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator);



}

bool FixLJCut::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillEps = [] (float a, float b) {
        return sqrt(a*b);
    };

    auto fillSig = [] (float a, float b) {
        return (a+b) / 2.0;
    };
    auto fillRCut = [this] (float a, float b) {
        return (float) std::fmax(a, b);
    };
    auto none = [] (float a){};

    auto fillRCutDiag = [this] () {
        return (float) state->rCut;
    };

    auto processEps = [] (float a) {
        return 24*a;
    };
    auto processSig = [] (float a) {
        return pow(a, 6);
    };
    auto processRCut = [] (float a) {
        return a*a;
    };
    prepareParameters(epsHandle, fillEps, processEps, false);
    prepareParameters(sigHandle, fillSig, processSig, false);
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);
    sendAllToDevice();
    return true;
}

string FixLJCut::restartChunk(string format) {
    stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}

bool FixLJCut::readFromRestart(pugi::xml_node restData) {
    cout << "Reading form restart" << endl;
    auto curr_param = restData.first_child();
    while (curr_param) {
        if (curr_param.name() == "parameter") {
            vector<float> val;
            string paramHandle = curr_param.attribute("handle").value();
            string s;
            istringstream ss(curr_param.value());
            while (ss >> s) {
                val.push_back(atof(s.c_str()));
            }
            initializeParameters(paramHandle, val);
        }
        curr_param = curr_param.next_sibling();
    }
    cout << "Reading LJ parameters from restart" << endl;
    return true;
}

bool FixLJCut::postRun() {

    return true;
}

void FixLJCut::addSpecies(string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);

}

vector<float> FixLJCut::getRCuts() { 
    vector<float> res;
    vector<float> &src = *(paramMap[rCutHandle]);
    for (float x : src) {
        if (x == DEFAULT_FILL) {
            res.push_back(-1);
        } else {
            res.push_back(x);
        }
    }

    return res;
}

void export_FixLJCut() {
    py::class_<FixLJCut, boost::shared_ptr<FixLJCut>, py::bases<FixPair>, boost::noncopyable > (
        "FixLJCut",
        py::init<boost::shared_ptr<State>, string> (py::args("state", "handle"))
    );

}
