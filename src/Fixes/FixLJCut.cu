#include "hip/hip_runtime.h"
#include "FixLJCut.h"

#include "BoundsGPU.h"
#include "GridGPU.h"
#include "list_macro.h"
#include "PairEvaluateIso.h"
#include "State.h"
#include "cutils_func.h"
const std::string LJCutType = "LJCut";

__device__ void eval (float3 &forceSum, float3 dr, float *params, float lenSqr, float multiplier) {
    printf("here\n");
    float rCutSqr = params[2];
    if (lenSqr < rCutSqr) {
        float epstimes24 = params[0];
        float sig6 = params[1];
        float p1 = epstimes24*2*sig6*sig6;
        float p2 = epstimes24*sig6;
        float r2inv = 1/lenSqr;
        float r6inv = r2inv*r2inv*r2inv;
        float forceScalar = r6inv * r2inv * (p1 * r6inv - p2) * multiplier;

        float3 forceVec = dr * forceScalar;
        forceSum += forceVec;
    }
}


FixLJCut::FixLJCut(boost::shared_ptr<State> state_, std::string handle_)
  : FixPair(state_, handle_, "all", LJCutType, true, 1),
    epsHandle("eps"), sigHandle("sig"), rCutHandle("rCut")
{
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);
    paramOrder = {epsHandle, sigHandle, rCutHandle};
}

void FixLJCut::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

   SAFECALL(( compute_force_iso<EvaluatorLJ, 3> <<<NBLOCK(nAtoms), PERBLOCK, 3*numTypes*numTypes*sizeof(float)>>>(
            nAtoms, gpd.xs(activeIdx), gpd.fs(activeIdx),
            neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(),
            state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU,
            neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator)
            ));

}

void FixLJCut::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int numTypes = state->atomParams.numTypes;
    GPUData &gpd = state->gpd;
    GridGPU &grid = state->gridGPU;
    int activeIdx = gpd.activeIdx();
    uint16_t *neighborCounts = grid.perAtomArray.d_data.data();
    float *neighborCoefs = state->specialNeighborCoefs;

    compute_energy_iso<EvaluatorLJ, 3><<<NBLOCK(nAtoms), PERBLOCK, 3*numTypes*numTypes*sizeof(float)>>>(nAtoms, gpd.xs(activeIdx), perParticleEng, 
                                                                                                        neighborCounts, grid.neighborlist.data(), grid.perBlockArray.d_data.data(), state->devManager.prop.warpSize, paramsCoalesced.data(), numTypes, state->boundsGPU, neighborCoefs[0], neighborCoefs[1], neighborCoefs[2], evaluator);



}

bool FixLJCut::prepareForRun() {
    //loop through all params and fill with appropriate lambda function, then send all to device
    auto fillEps = [] (float a, float b) {
        return sqrt(a*b);
    };

    auto fillSig = [] (float a, float b) {
        return (a+b) / 2.0;
    };
    auto fillRCut = [this] (float a, float b) {
        return (float) std::fmax(a, b);
    };
    auto none = [] (float a){};

    auto fillRCutDiag = [this] () {
        return (float) state->rCut;
    };

    auto processEps = [] (float a) {
        return 24*a;
    };
    auto processSig = [] (float a) {
        return pow(a, 6);
    };
    auto processRCut = [] (float a) {
        return a*a;
    };
    prepareParameters(epsHandle, fillEps, processEps, false);
    prepareParameters(sigHandle, fillSig, processSig, false);
    prepareParameters(rCutHandle, fillRCut, processRCut, true, fillRCutDiag);
    sendAllToDevice();
    return true;
}

std::string FixLJCut::restartChunk(std::string format) {
    std::stringstream ss;
    ss << restartChunkPairParams(format);
    return ss.str();
}

bool FixLJCut::readFromRestart(pugi::xml_node restData) {
    std::cout << "Reading form restart" << std::endl;
    auto curr_param = restData.first_child();
    while (curr_param) {
        if (curr_param.name() == "parameter") {
            std::vector<float> val;
            std::string paramHandle = curr_param.attribute("handle").value();
            std::string s;
            std::istringstream ss(curr_param.value());
            while (ss >> s) {
                val.push_back(atof(s.c_str()));
            }
            initializeParameters(paramHandle, val);
        }
        curr_param = curr_param.next_sibling();
    }
    std::cout << "Reading LJ parameters from restart" << std::endl;
    return true;
}

bool FixLJCut::postRun() {
    resetToPreproc(sigHandle);
    resetToPreproc(epsHandle);
    resetToPreproc(rCutHandle);

    return true;
}

void FixLJCut::addSpecies(std::string handle) {
    initializeParameters(epsHandle, epsilons);
    initializeParameters(sigHandle, sigmas);
    initializeParameters(rCutHandle, rCuts);

}

std::vector<float> FixLJCut::getRCuts() {  // to be called after prepare.  These are squares now
    return LISTMAP(float, float, rc, rCuts, sqrt(rc));
}

void export_FixLJCut() {
    boost::python::class_<FixLJCut,
                          boost::shared_ptr<FixLJCut>,
                          boost::python::bases<FixPair>, boost::noncopyable > (
        "FixLJCut",
        boost::python::init<boost::shared_ptr<State>, std::string> (
            boost::python::args("state", "handle"))
    );

}
