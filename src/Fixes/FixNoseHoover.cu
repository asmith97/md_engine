#include "hip/hip_runtime.h"
#include "FixNoseHoover.h"

#include <cmath>
#include <string>

#include <boost/python.hpp>

#include "cutils_func.h"
#include "Logging.h"
#include "State.h"

namespace py = boost::python;

std::string NoseHooverType = "NoseHoover";

// CUDA function to calculate the total kinetic energy

// CUDA function to rescale particle velocities
__global__ void rescale_cu(int nAtoms, uint groupTag, float4 *vs, float4 *fs, float scale)
{
    int idx = GETIDX();
    if (idx < nAtoms) {
        uint groupTagAtom = ((uint *) (fs+idx))[3];
        if (groupTag & groupTagAtom) {
            float4 vel = vs[idx];
            float invmass = vel.w;
            vel *= scale;
            vel.w = invmass;
            vs[idx] = vel;
        }
    }
}

FixNoseHoover::FixNoseHoover(boost::shared_ptr<State> state, std::string handle,
                             std::string groupHandle, float temp, float timeConstant)
        : Fix(state,
              handle,           // Fix handle
              groupHandle,      // Group handle
              NoseHooverType,   // Fix name
              false,            // forceSingle
              1                 // applyEvery
             ), temp(temp), frequency(1.0 / timeConstant),
                kineticEnergy(GPUArrayGlobal<float>(2)),
                ke_current(0.0), ndf(0),
                chainLength(3), nTimesteps(1), n_ys(1),
                weight(std::vector<double>(n_ys,1.0)),
                thermPos(std::vector<double>(chainLength,0.0)),
                thermVel(std::vector<double>(chainLength,0.0)),
                thermForce(std::vector<double>(chainLength,0.0)),
                thermMass(std::vector<double>(chainLength,0.0))
{

}

bool FixNoseHoover::prepareForRun()
{
    // Calculate current kinetic energy
    calculateKineticEnergy();
    updateMasses();

    return true;
}

bool FixNoseHoover::stepInit()
{
    return halfStep(true);
}

bool FixNoseHoover::stepFinal()
{
    return halfStep(false);
}

bool FixNoseHoover::halfStep(bool firstHalfStep)
{
    if (chainLength == 0) {
        mdWarning("Call of FixNoseHoover with zero thermostats in "
                  "the Nose-Hoover chain.");
        return false;
    }

    //! \todo Until now, we assume Boltzmann-constant = 1.0. Consider allowing
    //!       other units.
    double boltz = 1.0;

    // Update the desired temperature
    if (firstHalfStep) {
        if (updateTemperature()) {
            updateMasses();
        }
    }

    double scale = 1.0;

    // Get the total kinetic energy
    if (!firstHalfStep) {
        //! \todo This optimization assumes that the velocities are not changed
        //!       between stepFinal() and stepInit(). Can we add a check to make
        //!       sure this is indeed the case?
        calculateKineticEnergy();
    }

    // Equipartition at desired temperature
    double nkt = ndf * boltz * temp;

    // Update the forces
    thermForce.at(0) = (ke_current - nkt) / thermMass.at(0);

    // Multiple timestep procedure
    for (size_t i = 0; i < nTimesteps; ++i) {
        for (size_t j = 0; j < n_ys; ++j) {
            double timestep = weight.at(j)*state->dt / nTimesteps;
            double timestep2 = 0.5*timestep;
            double timestep4 = 0.25*timestep;
            double timestep8 = 0.125*timestep;

            // Update thermostat forces
            //! \todo Consider shorter thermostat chains
            thermForce.at(chainLength-1) =
                (
                    thermMass.at(chainLength-2) *
                    thermVel.at(chainLength-2) *
                    thermVel.at(chainLength-2) - boltz*temp
                ) / thermMass.at(chainLength-1);

            // Update thermostat velocities
            thermVel.back() += timestep4*thermForce.back();
            for (size_t k = chainLength-2; k > 0; --k) {
                double preFactor = std::exp( -timestep8*thermVel.at(k+1) );
                thermVel.at(k) *= preFactor;
                thermForce.at(k) = (
                        thermMass.at(k-1) *
                        thermVel.at(k-1) *
                        thermVel.at(k-1) - boltz*temp
                    ) / thermMass.at(k);
                thermVel.at(k) += timestep4 * thermForce.at(k);
                thermVel.at(k) *= preFactor;
            }

            double preFactor = std::exp( -timestep8*thermVel.at(1) );
            thermVel.at(0) *= preFactor;
            thermVel.at(0) += timestep4*thermForce.at(0);
            thermVel.at(0) *= preFactor;

            // Update particle velocities
            double scaleFactor = std::exp( -timestep2*thermVel.at(0) );
            scale *= scaleFactor;

            ke_current *= scaleFactor*scaleFactor;

            // Update the thermostat positions
            for (size_t k = 0; k < chainLength; ++k) {
                thermPos.at(k) += timestep2*thermVel.at(k);
            }

            // Update the forces
            thermVel.at(0) *= preFactor;
            thermForce.at(0) = (ke_current - nkt) / thermMass.at(0);
            thermVel.at(0) += timestep4 * thermForce.at(0);
            thermVel.at(0) *= preFactor;

            // Update thermostat velocities
            for (size_t k = 1; k < chainLength-1; ++k) {
                preFactor = std::exp( -timestep8*thermVel.at(k+1) );
                thermVel.at(k) *= preFactor;
                thermForce.at(k) = (
                        thermMass.at(k-1) *
                        thermVel.at(k-1) *
                        thermVel.at(k-1) - boltz*temp
                    ) / thermMass.at(k);
                thermVel.at(k) += timestep4 * thermForce.at(k);
                thermVel.at(k) *= preFactor;
            }

            thermForce.at(chainLength-1) = (
                    thermMass.at(chainLength-2) *
                    thermVel.at(chainLength-2) *
                    thermVel.at(chainLength-2) - boltz*temp
                ) / thermMass.at(chainLength-1);
            thermVel.at(chainLength-1) += timestep4*thermForce.at(chainLength-1);
        }
    }

    // Update particle velocites
    // scale gets converted to float, losing precision
    rescale(scale);

    return true;
}

bool FixNoseHoover::updateTemperature()
{
    // This should be modified to allow for temperature changes
    double newTemp = temp;

    if (temp != newTemp) {
        // Temperature changed
        temp = newTemp;
        return true;
    }

    // Temperature remained unchanged
    return false;
}

void FixNoseHoover::updateMasses()
{
    double boltz = 1.0;

    thermMass.at(0) = ndf * boltz * temp / (frequency*frequency);
    for (size_t i = 1; i < chainLength; ++i) {
        thermMass.at(i) = boltz*temp / (frequency*frequency);
    }
}

void FixNoseHoover::calculateKineticEnergy()
{
    size_t nAtoms = state->atoms.size();
    kineticEnergy.d_data.memset(0);
    SAFECALL((sumVectorSqr3DTagsOverW<float, float4>
        <<<NBLOCK(nAtoms), PERBLOCK, PERBLOCK*sizeof(float)>>>(
                kineticEnergy.d_data.data(),
                state->gpd.vs.getDevData(),
                nAtoms,
                groupTag,
                state->gpd.fs.getDevData(),
                state->devManager.prop.warpSize
        )));
    kineticEnergy.dataToHost();
    hipDeviceSynchronize();

    ke_current = kineticEnergy.h_data[0];
    ndf = *((int *) (kineticEnergy.h_data.data()+1));
    if (state->is2d) {
        ndf *= 2;
    } else {
        ndf *= 3;
    }
}

void FixNoseHoover::rescale(float scale)
{
    size_t nAtoms = state->atoms.size();
    rescale_cu<<<NBLOCK(nAtoms), PERBLOCK>>>(nAtoms,
                                             groupTag,
                                             state->gpd.vs.getDevData(),
                                             state->gpd.fs.getDevData(),
                                             scale);
}

void export_FixNoseHoover()
{
    py::class_<FixNoseHoover,                    // Class
               boost::shared_ptr<FixNoseHoover>, // HeldType
               py::bases<Fix>,                   // Base class
               boost::noncopyable>
    (
        "FixNoseHoover",
        py::init<boost::shared_ptr<State>, std::string, std::string, float, float>(
            py::args("state", "handle", "groupHandle", "temp", "timeConstant")
        )
    )
    ;
}
