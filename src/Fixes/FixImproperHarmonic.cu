#include "hip/hip_runtime.h"
#include "helpers.h"
#include "FixImproperHarmonic.h"
#include "FixHelpers.h"
#include "cutils_func.h"
#define SMALL 0.001f
#include "ImproperEvaluate.h"
namespace py = boost::python;
using namespace std;

const std::string improperHarmonicType = "ImproperHarmonic";
/*
__global__ void compute_cu(int nAtoms, float4 *xs, float4 *forces, hipTextureObject_t idToIdxs, ImproperGPU *impropers, int *startstops, BoundsGPU bounds, ImproperHarmonicType *parameters, int nParameters) {


    int idx = GETIDX();
    extern __shared__ int all_shr[];
    int idxBeginCopy = startstops[blockDim.x*blockIdx.x];
    int idxEndCopy = startstops[min(nAtoms, blockDim.x*(blockIdx.x+1))];

    ImproperGPU *impropers_shr = (ImproperGPU *) all_shr;
    ImproperHarmonicType *parameters_shr = (ImproperHarmonicType *) (impropers_shr + (idxEndCopy - idxBeginCopy));
    copyToShared<ImproperGPU>(impropers + idxBeginCopy, impropers_shr, idxEndCopy - idxBeginCopy);
    copyToShared<ImproperHarmonicType>(parameters, parameters_shr, nParameters);

    __syncthreads();
    if (idx < nAtoms) { //HEY - THIS SHOULD BE < nAtoms
  //      printf("going to compute %d\n", idx);
        int startIdx = startstops[idx];
        int endIdx = startstops[idx+1];
        //so start/end is the index within the entire bond list.
        //startIdx - idxBeginCopy gives my index in shared memory
        int shr_idx = startIdx - idxBeginCopy;
        int n = endIdx - startIdx;
        if (n) {
            int myIdxInImproper = impropers_shr[shr_idx].type >> 29;
            int idSelf = impropers_shr[shr_idx].ids[myIdxInImproper];
            
            int idxSelf = tex2D<int>(idToIdxs, XIDX(idSelf, sizeof(int)), YIDX(idSelf, sizeof(int)));
        
            float3 pos = make_float3(xs[idxSelf]);
           // printf("I am idx %d and I am evaluating atom with pos %f %f %f\n", idx, pos.x, pos.y, pos.z);
            float3 forceSum = make_float3(0, 0, 0);
            for (int i=0; i<n; i++) {
                ImproperGPU improper = impropers_shr[shr_idx + i];
                uint32_t typeFull = improper.type;
                myIdxInImproper = typeFull >> 29;
                int type = static_cast<int>((typeFull << 3) >> 3);   
                ImproperHarmonicType improperType = parameters_shr[type];
                float3 positions[4];
                positions[myIdxInImproper] = pos;
                int toGet[3];
                if (myIdxInImproper==0) {
                    toGet[0] = 1;
                    toGet[1] = 2;
                    toGet[2] = 3;
                } else if (myIdxInImproper==1) {
                    toGet[0] = 0;
                    toGet[1] = 2;
                    toGet[2] = 3;
                } else if (myIdxInImproper==2) {
                    toGet[0] = 0;
                    toGet[1] = 1;
                    toGet[2] = 3;
                } else if (myIdxInImproper==3) {
                    toGet[0] = 0;
                    toGet[1] = 1;
                    toGet[2] = 2;
                }
                for (int i=0; i<3; i++) {
                    positions[toGet[i]] = make_float3(perAtomFromId(idToIdxs, xs, improper.ids[toGet[i]]));
                }
                for (int i=1; i<3; i++) {
                    positions[i] = positions[0] + bounds.minImage(positions[i]-positions[0]);
                }
                float3 directors[3]; //vb_xyz in lammps
                float lenSqrs[3]; //bnmag2 in lammps
                float lens[3]; //bnmag in lammps
                float invLenSqrs[3]; //sb in lammps
                float invLens[3];
                directors[0] = positions[0] - positions[1];
                directors[1] = positions[2] - positions[1];
                directors[2] = positions[3] - positions[2];
                for (int i=0; i<3; i++) {
                    //printf("directors %d is %f %f %f\n", i, directors[i].x, directors[i].y, directors[i].z);
                    lenSqrs[i] = lengthSqr(directors[i]);
                    lens[i] = sqrtf(lenSqrs[i]);
                    invLenSqrs[i] = 1.0f / lenSqrs[i];
                    invLens[i] = 1.0f / lens[i];
                 //   printf("inv len sqrs %d is %f\n", i, invLenSqrs[i]);
                }

                float angleBits[3]; //c0, 1, 2
                angleBits[0] = dot(directors[0], directors[2]) * invLens[0] * invLens[2];
                angleBits[1] = dot(directors[0], directors[1]) * invLens[0] * invLens[1];
                angleBits[2] = -dot(directors[2], directors[1]) * invLens[2] * invLens[1];

                float scValues[3]; //???, is s1, s2, s12 in lammps
                for (int i=0; i<2; i++) {
                    scValues[i] = 1.0f - angleBits[i+1] * angleBits[i+1];
                    if (scValues[i] < SMALL) {
                        scValues[i] = SMALL;
                    }
                    scValues[i] = 1.0 / scValues[i];
                }
                scValues[2] = sqrtf(scValues[0] * scValues[1]);
                float c = (angleBits[1]*angleBits[2] + angleBits[0]) * scValues[2];

                if (c > 1.0f) {
                    c = 1.0f;
                } else if (c < -1.0f) {
                    c = -1.0f;
                }
                float s = sqrtf(1.0f - c*c);
                if (s < SMALL) {
                    s = SMALL;
                }
                float dTheta = acosf(c) - improperType.thetaEq;

                float a = improperType.k * dTheta;
                a *= -2.0f / s;
                scValues[2] *= a;
                c *= a;
                float a11 = c * invLenSqrs[0] * scValues[0];
                float a22 = - invLenSqrs[1] * (2.0f * angleBits[0] * scValues[2] - c * (scValues[0] + scValues[1]));
                float a33 = c * invLenSqrs[2] * scValues[1];
                float a12 = -invLens[0] * invLens[1] * (angleBits[1] * c * scValues[0] + angleBits[2] * scValues[2]);
                float a13 = -invLens[0] * invLens[2] * scValues[2];
                float a23 = invLens[1] * invLens[2] * (angleBits[2] * c * scValues[1] + angleBits[1] * scValues[2]);

                float3 myForce = make_float3(0, 0, 0);
                float3 sFloat3 = make_float3(
                        a22*directors[1].x + a23*directors[2].x + a12*directors[0].x
                        ,  a22*directors[1].y + a23*directors[2].y + a12*directors[0].y
                        ,  a22*directors[1].z + a23*directors[2].z + a12*directors[0].z
                        );
                if (myIdxInImproper <= 1) {
                    float3 a11Dir1 = directors[0] * a11;
                    float3 a12Dir2 = directors[1] * a12;
                    float3 a13Dir3 = directors[2] * a13;
                    myForce.x += a11Dir1.x + a12Dir2.x + a13Dir3.x;
                    myForce.y += a11Dir1.y + a12Dir2.y + a13Dir3.y;
                    myForce.z += a11Dir1.z + a12Dir2.z + a13Dir3.z;

                    if (myIdxInImproper == 1) {
                        
                        myForce = -sFloat3 - myForce;
                    }
                  //      printf("improper idx 1 gets force %f %f %f\n", myForce.x, myForce.y, myForce.z);
                 //   } else {
                   //     printf("improper idx 0 gets force %f %f %f\n", myForce.x, myForce.y, myForce.z);
                  //  }
                } else {
                    float3 a13Dir1 = directors[0] * a13;
                    float3 a23Dir2 = directors[1] * a23;
                    float3 a33Dir3 = directors[2] * a33;
                    myForce.x += a13Dir1.x + a23Dir2.x + a33Dir3.x;
                    myForce.y += a13Dir1.y + a23Dir2.y + a33Dir3.y;
                    myForce.z += a13Dir1.z + a23Dir2.z + a33Dir3.z;
                    if (myIdxInImproper == 2) {
                        myForce = sFloat3 - myForce;
                   //     printf("improper idx 2 gets force %f %f %f\n", myForce.x, myForce.y, myForce.z);
                    }

                   // } else {
                   //     printf("improper idx 3 gets force %f %f %f\n", myForce.x, myForce.y, myForce.z);
                  //  }


                }
                forceSum += myForce;


            }
            forces[idxSelf] += forceSum;
        }
    }
}
*/

FixImproperHarmonic::FixImproperHarmonic(SHARED(State) state_, string handle)
    : FixPotentialMultiAtom (state_, handle, improperHarmonicType, true),
      pyListInterface(&forcers, &pyForcers) {}


void FixImproperHarmonic::compute(bool computeVirials) {
    int nAtoms = state->atoms.size();
    int activeIdx = state->gpd.activeIdx();
    compute_force_improper<<<NBLOCK(nAtoms), PERBLOCK, sizeof(ImproperGPU) * maxForcersPerBlock + forcers.size() * sizeof(ImproperHarmonicType)>>>(nAtoms, state->gpd.xs(activeIdx), state->gpd.fs(activeIdx), state->gpd.idToIdxs.getTex(), forcersGPU.data(), forcerIdxs.data(), state->boundsGPU, parameters.data(), parameters.size(), evaluator);

}
void FixImproperHarmonic::singlePointEng(float *perParticleEng) {
    int nAtoms = state->atoms.size();
    int activeIdx = state->gpd.activeIdx();
    compute_energy_improper<<<NBLOCK(nAtoms), PERBLOCK, sizeof(ImproperGPU) * maxForcersPerBlock + forcers.size() * sizeof(ImproperHarmonicType)>>>(nAtoms, state->gpd.xs(activeIdx), perParticleEng, state->gpd.idToIdxs.getTex(), forcersGPU.data(), forcerIdxs.data(), state->boundsGPU, parameters.data(), parameters.size(), evaluator);

}

void FixImproperHarmonic::createImproper(Atom *a, Atom *b, Atom *c, Atom *d, double k, double thetaEq, int type) {
    vector<Atom *> atoms = {a, b, c, d};
    validAtoms(atoms);
    if (type == -1) {
        assert(k!=COEF_DEFAULT and thetaEq!=COEF_DEFAULT);
    }
    forcers.push_back(ImproperHarmonic(a, b, c, d, k, thetaEq, type));
    pyListInterface.updateAppendedMember();
}
void FixImproperHarmonic::setImproperTypeCoefs(int type, double k, double thetaEq) {
    assert(thetaEq>=0);
    ImproperHarmonic dummy(k, thetaEq, type);
    setForcerType(type, dummy);
}




string FixImproperHarmonic::restartChunk(string format) {
    stringstream ss;

    return ss.str();
}

void export_FixImproperHarmonic() {

    boost::python::class_<FixImproperHarmonic,
                          SHARED(FixImproperHarmonic),
                          boost::python::bases<Fix, TypedItemHolder> > (
        "FixImproperHarmonic",
        boost::python::init<SHARED(State), string> (
                boost::python::args("state", "handle"))
    )
    .def("createImproper", &FixImproperHarmonic::createImproper,
            (boost::python::arg("k")=COEF_DEFAULT,
             boost::python::arg("thetaEq")=COEF_DEFAULT,
             boost::python::arg("type")=-1)
        )
    .def("setImproperTypeCoefs", &FixImproperHarmonic::setImproperTypeCoefs,
            (boost::python::arg("type")=COEF_DEFAULT,
             boost::python::arg("k")=COEF_DEFAULT,
             boost::python::arg("thetaEq")=COEF_DEFAULT
             )
        )
    ;

}

